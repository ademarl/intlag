#include "hip/hip_runtime.h"

//------------------------------------------------------------------------------
// Copyright (c) 2014 Ademar Marques Lacerda Filho
//
// This Source Code Form is subject to the terms of the Mozilla Public
// License, v. 2.0. If a copy of the MPL was not distributed with this
// file, You can obtain one at http://mozilla.org/MPL/2.0/.
//------------------------------------------------------------------------------


#include <math.h>
#include "cuda_min_max.h"

namespace intlag {

__device__ inline
float min(float a, float b)
{
  return fminf(a,b);
}

__device__ inline
double min(double a, double b)
{
  return fmin(a,b);
}

__device__ inline
float max(float a, float b)
{
  return fmaxf(a,b);
}

__device__ inline
double max(double a, double b)
{
  return fmax(a,b);
}

__device__ inline
float min(float a, float b, float c, float d)
{
  return fminf(fminf(a,b), fminf(c,d));
}

__device__ inline
double min(double a, double b, double c, double d)
{
  return fmin(fmin(a,b), fmin(c,d));
}

__device__ inline
float max(float a, float b, float c, float d)
{
  return fmaxf(fmaxf(a,b), fmaxf(c,d));
}

__device__ inline
double max(double a, double b, double c, double d)
{
  return fmax(fmax(a,b), fmax(c,d));
}


} // namespace intlag



