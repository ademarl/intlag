#include "hip/hip_runtime.h"
#include <iostream>
#include <cstdio>
#include <cstdlib>

#include "cuda_interval_lib.h"
#include "helper_timer.h"

#include<omp.h>
#include <boost/numeric/interval.hpp>
using boost::numeric::interval;
using namespace boost::numeric;

//#include <hip/hip_runtime.h>
//#include <hip/hip_runtime.h>
//#include "hip/hip_runtime_api.h"

using namespace std;


typedef interval_gpu<double> I_t;

// Kernel
//template <class T>
__global__ void ADD(I_t *a, I_t *b, I_t *c, int N)
{
	int i = blockIdx.x*blockDim.x +threadIdx.x;
	if (i < N)
		c[i] = a[i] + b[i];
}

// Auxiliar
void printI(I_t x);
void gpu_sum(I_t *a, I_t *b, I_t *c, int N);
void read_intervals(I_t *a, I_t *b, int N);

template<class T>
void read_intervals(interval<T> *x,interval<T> *y, int N);

template<class T>
void printI(interval<T> x){
	cout << "[" << lower(x) << ", " << upper(x) << "]" << endl;
}



int main(int argc,char *argv[])
{
	int method = 0;
	int iter = 100;
	bool pflag = false;

	switch(argc){

	case 1:
		break;

	case 4:
		if(!strcmp(argv[3], "-p"))
			pflag = true;

	case 3:
		iter = atoi(argv[2]);

	case 2:
		method = atoi(argv[1]);
		break;

	default:
		method = atoi(argv[1]);
		iter = atoi(argv[2]);
		if(!strcmp(argv[3], "-p"))
			pflag = true;
	}

	int N;
	I_t *a = NULL, *b = NULL, *c = NULL;
	interval<double> *x, *y, *z;

	cin >> N;
	N = N/2;
	if (method >= 2){
		a = new I_t[N];
		b = new I_t[N];
		c = new I_t[N];
		read_intervals(a, b, N);
	}
	else{ 
		x = new interval<double>[N];
		y = new interval<double>[N];
		z = new interval<double>[N];
		read_intervals(x, y, N);
	}
	
	// Run "iter" times and clock the time	
	StopWatchInterface *timer;
	sdkCreateTimer(&timer);
	sdkResetTimer(&timer);
	sdkStartTimer(&timer);
	for (int i = 0; i < iter; ++i){
		if (method >= 2){
			gpu_sum(a, b, c, N);
			hipDeviceSynchronize();
		}
		else if (method == 1){
			omp_set_num_threads(8);
			#pragma omp parallel for schedule(dynamic, 500)
			for (int j = 0; j < N; ++j)
				z[j] = x[j] + y[j];
		}
		else{
			for (int j = 0; j < N; ++j)
				z[j] = x[j] + y[j];
		}
	}
	sdkStopTimer(&timer);

	// print if user asked for it
	if(pflag){
		for (int i = 0; i < N; ++i){
			if (method >= 2)
				printI(c[i]);
			else
				printI(z[i]);
		}
	}
	
	// print execution time
	printf ("Time for the execution of %d interval sums %d times: %f s\n", N, iter, sdkGetTimerValue(&timer)/1000);

	if (method >= 2){
		delete[] a;
		delete[] b;
		delete[] c;
	}
	else {
		delete[] x;
		delete[] y;
		delete[] z;
	}

	return 0;
}

void gpu_sum(I_t* a, I_t* b, I_t* c, int N){

	I_t *d_a, *d_b, *d_c;
	// test malloc time impact (no impact)
	//interval_gpu<double> *z = (interval_gpu<double> *) malloc(N*sizeof(I_t));

	hipMalloc((void **) &d_a, N*sizeof(I_t));
	hipMalloc((void **) &d_b, N*sizeof(I_t));
	hipMalloc((void **) &d_c, N*sizeof(I_t));

	hipMemcpy(d_a, a, N*sizeof(I_t), hipMemcpyHostToDevice);
	hipMemcpy(d_b, b, N*sizeof(I_t), hipMemcpyHostToDevice);

	// how to set the number of blocks and threads?
	ADD<<<1024, (N+1024)/1024>>>(d_a, d_b, d_c, N);
	//hipDeviceSynchronize();

	hipMemcpy(c, d_c, N*sizeof(I_t), hipMemcpyDeviceToHost);

	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_c);
}

void printI(I_t x){
	cout << "[" << x.lower() << ", " << x.upper() << "]" << endl;
}


void read_intervals(I_t* a, I_t* b, int N){

	double aux, aux2;

	for (int i = 0; i < N; ++i){
		cin >> aux; cin >> aux2;
		a[i] = I_t(aux, aux2);
	}
	for (int i = 0; i < N; ++i){
		cin >> aux; cin >> aux2;
		b[i] = I_t(aux, aux2);
	}

}

template<class T>
void read_intervals(interval<T> *x,interval<T> *y, int N){

	double aux, aux2;

	for (int i = 0; i < N; ++i){
		cin >> aux, cin >> aux2;
		x[i] = interval<double>(aux, aux2);
	}
	for (int i = 0; i < N; ++i){
		cin >> aux, cin >> aux2;
		y[i] = interval<double>(aux, aux2);
	}

}

