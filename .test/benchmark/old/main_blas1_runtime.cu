

// This program intends to run performance tests box_gpu's BLAS 1
// input must contain an alpha, followed by the number of intervals and the intervals themselves (lower first)

#include <iostream>

#include "io_aux.h"
#include "blas1_runtime.h"


// NOTE: Double precision only
int main(int argc, char** argv) {

	std::cout << "Runtime test cases for interval BLAS level 1" << endl;

	int method = 0;
	int n_iter = 100;
	set_options(argc, argv, &method, &n_iter);

	switch(method) {
		case 0:
				serial_blas1_runtime(n_iter);
			break;
		case 1:
				openmp_blas1_runtime(n_iter);
			break;
		case 2:
				cuda_blas1_runtime(n_iter);
			break;
		
		default:
			std::cerr << "Error: Method undefined, 0-2 expected."<< endl;
			exit(1);
	}

	return 0;
}
