#include "hip/hip_runtime.h"

//------------------------------------------------------------------------------
// Copyright (c) 2014 Ademar Marques Lacerda Filho
//
// This Source Code Form is subject to the terms of the Mozilla Public
// License, v. 2.0. If a copy of the MPL was not distributed with this
// file, You can obtain one at http://mozilla.org/MPL/2.0/.
//------------------------------------------------------------------------------


#ifndef BOOST_BENCH_H
#define BOOST_BENCH_H

#include "boost/numeric/interval.hpp"
using boost::numeric::interval;
using namespace boost::numeric;


#include "aux/case.h"
#include "aux/reference.h"
#include "../include/blas/boost_serial_blas.h"

#include "gtest/gtest.h"



namespace intlag {
namespace bench {

template<class T, class U>
void acopy(int n, intlag::Interval<T> const *x, interval<U> *y) {
		for(int i = 0; i < n; ++i)
			y[i] = interval<T>(x[i].inf(), x[i].sup());
}

//----------------------- Fixture --------------------------------------------//

class BoostBench : public BenchTest {
  public:

    BoostBench() {
      r = Reference::getInstance();
    }
    virtual ~BoostBench() {}

    void SetUp() {}
    void TearDown() {}

    Reference* r;
};


//----------------------- Scal Bench -----------------------------------------//

template <class T>
class BoostBenchScal : public BoostBench  {
   public:

    void begin() {


      n = r->length;
      alpha = interval<T>((r->alpha).inf(), (r->alpha).sup());
      x = new interval<T> [n];
      acopy(n, r->x, x);
    }

    void run() {
      bi::scal(n, alpha, x);
    }

    void end() {
      delete[] x;
    }

    int n;
    interval<T> *x, alpha;
};
BENCH_F_F(BoostBench, Scal, BoostBenchScal)


//----------------------- AXPY Bench -----------------------------------------//

template <class T>
class BoostBenchAXPY : public BoostBench  {
   public:

    void begin() {
      n = r->length;
      alpha = interval<T>((r->alpha).inf(), (r->alpha).sup());
      x = new interval<T> [n];
      y = new interval<T> [n];
      acopy(n, r->x, x);
      acopy(n, r->y, y);
    }

    void run() {
      bi::axpy(n, alpha, x, y);
    }

    void end() {
      delete[] x;
      delete[] y;
    }

    int n;
    interval<T> *x, *y, alpha;
};
BENCH_F_F(BoostBench, AXPY, BoostBenchAXPY)


//----------------------- ASum Bench -----------------------------------------//

template <class T>
class BoostBenchASum : public BoostBench  {
   public:

    void begin() {
      n = r->length;
      x = new interval<T> [n];
      acopy(n, r->x, x);
    }

    void run() {
      bi::asum(n, &ret, x);
    }

    void end() {
      delete[] x;
    }

    int n;
    interval<T> *x, ret;
};
BENCH_F_F(BoostBench, Asum, BoostBenchASum)


//----------------------- Dot Bench -----------------------------------------//

template <class T>
class BoostBenchDot : public BoostBench  {
   public:

    void begin() {
      n = r->length;
      x = new interval<T> [n];
      y = new interval<T> [n];
      acopy(n, r->x, x);
      acopy(n, r->y, y);
    }

    void run() {
      bi::dot(n, &ret, x, y);
    }

    void end() {
      delete[] x;
      delete[] y;
    }

    int n;
    interval<T> *x, *y, ret;
};
BENCH_F_F(BoostBench, Dot, BoostBenchDot)


//----------------------- Norm2 Bench -----------------------------------------//

template <class T>
class BoostBenchNorm2 : public BoostBench  {
   public:

    void begin() {
      n = r->length;
      x = new interval<T> [n];
      acopy(n, r->x, x);
    }

    void run() {
      bi::norm2(n, &ret, x);
    }

    void end() {
      delete[] x;
    }

    int n;
    interval<T> *x, ret;
};
BENCH_F_F(BoostBench, Norm2, BoostBenchNorm2)


//----------------------- Rot Bench -----------------------------------------//

template <class T>
class BoostBenchRot : public BoostBench  {
   public:

    void begin() {
      n = r->length;
      x = new interval<T> [n];
      acopy(n, r->x, x);
      y = new interval<T> [n];
      acopy(n, r->y, y);
    }

    void run() {
      bi::rot(n, x, y, interval<T>(0.5), interval<T>(0.5));
    }

    void end() {
      delete[] x;
      delete[] y;
    }

    int n;
    interval<T> *x, *y;
};
BENCH_F_F(BoostBench, Rot, BoostBenchRot)


//----------------------- Rotm Bench -----------------------------------------//

template <class T>
class BoostBenchRotm : public BoostBench  {
   public:

    void begin() {
      h[0] = -1; h[1] = 0.5; h[2] = 0.6; h[3] = 0.7; h[4] = 0.8;
      n = r->length;
      x = new interval<T> [n];
      acopy(n, r->x, x);
      y = new interval<T> [n];
      acopy(n, r->y, y);
    }

    void run() {
      bi::rotm(n, x, y, h);
    }

    void end() {
      delete[] x;
      delete[] y;
    }

    int n;
    T h[5];
    interval<T> *x, *y;
};
BENCH_F_F(BoostBench, Rotm, BoostBenchRotm)


//----------------------- Ger Bench -----------------------------------------//


template <class T>
class BoostBenchGer : public BoostBench  {
   public:

    void begin() {
      n = r->length;
      dim = std::sqrt(n);
      alpha = interval<T>((r->alpha).inf(), (r->alpha).sup());
      A = new interval<T> [n];
      x = new interval<T> [dim];
      y = new interval<T> [dim];
      acopy(n, r->x, A);
      acopy(dim, r->x, x);
      acopy(dim, r->y, y);
    }

    void run() {
      bi::ger(dim, dim, alpha, x, y, A);
    }

    void end() {
      delete[] A;
      delete[] x;
      delete[] y;
    }

    int n, dim;
    interval<T> *A, *x, *y, alpha, beta;
};
BENCH_F_F(BoostBench, Ger, BoostBenchGer)


//----------------------- Syr Bench -----------------------------------------//


template <class T>
class BoostBenchSyr : public BoostBench  {
   public:

    void begin() {
      n = r->length;
      dim = std::sqrt(n);
      alpha = interval<T>((r->alpha).inf(), (r->alpha).sup());
      A = new interval<T> [n];
      x = new interval<T> [dim];
      acopy(n, r->x, A);
      acopy(dim, r->x, x);
    }

    void run() {
      bi::syr('u', dim, alpha, x, A);
    }

    void end() {
      delete[] A;
      delete[] x;
    }

    int n, dim;
    interval<T> *A, *x, alpha, beta;
};
BENCH_F_F(BoostBench, Syr, BoostBenchSyr)


//----------------------- Syr2 Bench -----------------------------------------//


template <class T>
class BoostBenchSyr2 : public BoostBench  {
   public:

    void begin() {
      n = r->length;
      dim = std::sqrt(n);
      alpha = interval<T>((r->alpha).inf(), (r->alpha).sup());
      A = new interval<T> [n];
      x = new interval<T> [dim];
      y = new interval<T> [dim];
      acopy(n, r->x, A);
      acopy(dim, r->x, x);
      acopy(dim, r->y, y);
    }

    void run() {
      bi::syr2('u', dim, alpha, x, y, A);
    }

    void end() {
      delete[] A;
      delete[] x;
      delete[] y;
    }

    int n, dim;
    interval<T> *A, *x, *y, alpha, beta;
};
BENCH_F_F(BoostBench, Syr2, BoostBenchSyr2)


//----------------------- Spr Bench -----------------------------------------//


template <class T>
class BoostBenchSpr : public BoostBench  {
   public:

    void begin() {
      n = r->length;
      dim = std::sqrt(n);
      alpha = interval<T>((r->alpha).inf(), (r->alpha).sup());
      A = new interval<T> [n];
      x = new interval<T> [dim];
      acopy(n, r->x, A);
      acopy(dim, r->x, x);
    }

    void run() {
      bi::spr('u', dim, alpha, x, A);
    }

    void end() {
      delete[] A;
      delete[] x;
    }

    int n, dim;
    interval<T> *A, *x, alpha, beta;
};
BENCH_F_F(BoostBench, Spr, BoostBenchSpr)


//----------------------- Spr2 Bench -----------------------------------------//


template <class T>
class BoostBenchSpr2 : public BoostBench  {
   public:

    void begin() {
      n = r->length;
      dim = std::sqrt(n);
      alpha = interval<T>((r->alpha).inf(), (r->alpha).sup());
      A = new interval<T> [n];
      x = new interval<T> [dim];
      y = new interval<T> [dim];
      acopy(n, r->x, A);
      acopy(dim, r->x, x);
      acopy(dim, r->y, y);
    }

    void run() {
      bi::spr2('u', dim, alpha, x, y, A);
    }

    void end() {
      delete[] A;
      delete[] x;
      delete[] y;
    }

    int n, dim;
    interval<T> *A, *x, *y, alpha, beta;
};
BENCH_F_F(BoostBench, Spr2, BoostBenchSpr2)


//----------------------- Gemv Bench -----------------------------------------//


template <class T>
class BoostBenchGemv : public BoostBench  {
   public:

    void begin() {
      n = r->length;
      dim = std::sqrt(n);
      alpha = interval<T>((r->alpha).inf(), (r->alpha).sup());
      beta = interval<T>((r->beta).inf(), (r->beta).sup());
      A = new interval<T> [n];
      x = new interval<T> [dim];
      y = new interval<T> [dim];
      acopy(n, r->x, A);
      acopy(dim, r->x, x);
      acopy(dim, r->y, y);
    }

    void run() {
      bi::gemv(dim, dim, alpha, beta, A, x, y);
    }

    void end() {
      delete[] A;
      delete[] x;
      delete[] y;
    }

    int n, dim;
    interval<T> *A, *x, *y, alpha, beta;
};
BENCH_F_F(BoostBench, Gemv, BoostBenchGemv)


//----------------------- Gbmv Bench -----------------------------------------//


template <class T>
class BoostBenchGbmv : public BoostBench  {
   public:

    void begin() {
      n = r->length;
      dim = std::sqrt(n);
      band = dim/4;
      alpha = interval<T>((r->alpha).inf(), (r->alpha).sup());
      beta = interval<T>((r->beta).inf(), (r->beta).sup());
      A = new interval<T> [n];
      x = new interval<T> [dim];
      y = new interval<T> [dim];
      acopy(n, r->x, A);
      acopy(dim, r->x, x);
      acopy(dim, r->y, y);
    }

    void run() {
      bi::gbmv(dim, dim, band, band, alpha, beta, A, x, y);
    }

    void end() {
      delete[] A;
      delete[] x;
      delete[] y;
    }

    int n, dim, band;
    interval<T> *A, *x, *y, alpha, beta;
};
BENCH_F_F(BoostBench, Gbmv, BoostBenchGbmv)


//----------------------- Symv Bench -----------------------------------------//


template <class T>
class BoostBenchSymv : public BoostBench  {
   public:

    void begin() {
      n = r->length;
      dim = std::sqrt(n);
      alpha = interval<T>((r->alpha).inf(), (r->alpha).sup());
      beta = interval<T>((r->beta).inf(), (r->beta).sup());
      A = new interval<T> [n];
      x = new interval<T> [dim];
      y = new interval<T> [dim];
      acopy(n, r->x, A);
      acopy(dim, r->x, x);
      acopy(dim, r->y, y);
    }

    void run() {
      bi::symv('u', dim, alpha, beta, A, x, y);
    }

    void end() {
      delete[] A;
      delete[] x;
      delete[] y;
    }

    int n, dim;
    interval<T> *A, *x, *y, alpha, beta;
};
BENCH_F_F(BoostBench, Symv, BoostBenchSymv)


//----------------------- Sbmv Bench -----------------------------------------//


template <class T>
class BoostBenchSbmv : public BoostBench  {
   public:

    void begin() {
      n = r->length;
      dim = std::sqrt(n);
      band = dim/4;
      alpha = interval<T>((r->alpha).inf(), (r->alpha).sup());
      beta = interval<T>((r->beta).inf(), (r->beta).sup());
      A = new interval<T> [n];
      x = new interval<T> [dim];
      y = new interval<T> [dim];
      acopy(n, r->x, A);
      acopy(dim, r->x, x);
      acopy(dim, r->y, y);
    }

    void run() {
      bi::sbmv('u', dim, band, alpha, beta, A, x, y);
    }

    void end() {
      delete[] A;
      delete[] x;
      delete[] y;
    }

    int n, dim, band;
    interval<T> *A, *x, *y, alpha, beta;
};
BENCH_F_F(BoostBench, Sbmv, BoostBenchSbmv)


//----------------------- Spmv Bench -----------------------------------------//


template <class T>
class BoostBenchSpmv : public BoostBench  {
   public:

    void begin() {
      n = r->length;
      dim = std::sqrt(n);
      alpha = interval<T>((r->alpha).inf(), (r->alpha).sup());
      beta = interval<T>((r->beta).inf(), (r->beta).sup());
      A = new interval<T> [n];
      x = new interval<T> [dim];
      y = new interval<T> [dim];
      acopy(n, r->x, A);
      acopy(dim, r->x, x);
      acopy(dim, r->y, y);
    }

    void run() {
      bi::spmv('u', dim, alpha, beta, A, x, y);
    }

    void end() {
      delete[] A;
      delete[] x;
      delete[] y;
    }

    int n, dim;
    interval<T> *A, *x, *y, alpha, beta;
};
BENCH_F_F(BoostBench, Spmv, BoostBenchSpmv)


//----------------------- Trmv Bench -----------------------------------------//


template <class T>
class BoostBenchTrmv : public BoostBench  {
   public:

    void begin() {
      n = r->length;
      dim = std::sqrt(n);
      A = new interval<T> [n];
      x = new interval<T> [dim];
      acopy(n, r->y, A);
      acopy(dim, r->x, x);
    }

    void run() {
      bi::trmv('u', dim, A, x);
    }

    void end() {
      delete[] A;
      delete[] x;
    }

    int n, dim;
    interval<T> *A, *x;
};
BENCH_F_F(BoostBench, Trmv, BoostBenchTrmv)


//----------------------- Tbmv Bench -----------------------------------------//


template <class T>
class BoostBenchTbmv : public BoostBench  {
   public:

    void begin() {
      n = r->length;
      dim = std::sqrt(n);
      band = dim/4;
      A = new interval<T> [n];
      x = new interval<T> [dim];
      acopy(n, r->y, A);
      acopy(dim, r->x, x);
    }

    void run() {
      bi::tbmv('u', dim, band, A, x);
    }

    void end() {
      delete[] A;
      delete[] x;
    }

    int n, dim, band;
    interval<T> *A, *x;
};
BENCH_F_F(BoostBench, Tbmv, BoostBenchTbmv)


//----------------------- Tpmv Bench -----------------------------------------//


template <class T>
class BoostBenchTpmv : public BoostBench  {
   public:

    void begin() {
      n = r->length;
      dim = std::sqrt(n);
      A = new interval<T> [n];
      x = new interval<T> [dim];
      acopy(n, r->y, A);
      acopy(dim, r->x, x);
    }

    void run() {
      bi::tpmv('u', dim, A, x);
    }

    void end() {
      delete[] A;
      delete[] x;
    }

    int n, dim;
    interval<T> *A, *x;
};
BENCH_F_F(BoostBench, Tpmv, BoostBenchTpmv)


//----------------------- Trsv Bench -----------------------------------------//


template <class T>
class BoostBenchTrsv : public BoostBench  {
   public:

    void begin() {
      n = r->length;
      dim = std::sqrt(n);
      A = new interval<T> [n];
      x = new interval<T> [dim];
      acopy(n, r->y, A);
      acopy(dim, r->x, x);
      for(int i = 0; i < dim; ++i)
        A[i*dim+i] = abs(A[i*dim+i]) + interval<T>(0.1);
    }

    void run() {
      bi::trsv('u', dim, A, x);
    }

    void end() {
      delete[] A;
      delete[] x;
    }

    int n, dim;
    interval<T> *A, *x;
};
BENCH_F_F(BoostBench, Trsv, BoostBenchTrsv)


//----------------------- Tbsv Bench -----------------------------------------//


template <class T>
class BoostBenchTbsv : public BoostBench  {
   public:

    void begin() {
      n = r->length;
      dim = std::sqrt(n);
      band = dim/4;
      A = new interval<T> [n];
      x = new interval<T> [dim];
      acopy(n, r->y, A);
      acopy(dim, r->x, x);
      for(int i = 0; i < dim; ++i)
        A[i*band+i] = abs(A[i*band+i]) + interval<T>(0.1);
    }

    void run() {
      bi::tbsv('u', dim, band, A, x);
    }

    void end() {
      delete[] A;
      delete[] x;
    }

    int n, dim, band;
    interval<T> *A, *x;
};
BENCH_F_F(BoostBench, Tbsv, BoostBenchTbsv)


//----------------------- Tpsv Bench -----------------------------------------//


template <class T>
class BoostBenchTpsv : public BoostBench  {
   public:

    void begin() {
      n = r->length;
      dim = std::sqrt(n);
      A = new interval<T> [n];
      x = new interval<T> [dim];
      acopy(n, r->y, A);
      acopy(dim, r->x, x);
      for(int i = 0; i < dim; ++i)
        A[INDEX_TRIAN_UP(dim, i, i)] = abs(A[INDEX_TRIAN_UP(dim, i, i)]) + interval<T>(0.1);
    }

    void run() {
      bi::tpsv('u', dim, A, x);
    }

    void end() {
      delete[] A;
      delete[] x;
    }

    int n, dim;
    interval<T> *A, *x;
};
BENCH_F_F(BoostBench, Tpsv, BoostBenchTpsv)


//----------------------- Gemm Bench -----------------------------------------//


template <class T>
class BoostBenchGemm : public BoostBench  {
   public:

    void begin() {
      n = r->length;
      dim = std::sqrt(n);
      alpha = interval<T>((r->alpha).inf(), (r->alpha).sup());
      beta = interval<T>((r->beta).inf(), (r->beta).sup());
      A = (interval<T>*) malloc(n*sizeof(interval<T>));
      B = (interval<T>*) malloc(n*sizeof(interval<T>));
      C = (interval<T>*) malloc(n*sizeof(interval<T>));
      acopy(n, r->x, A);
      acopy(n, r->y, B);
      acopy(n, r->y, C);
    }

    void run() {
      bi::gemm(dim, dim, dim, alpha, beta, A, B, C);
    }

    void end() {
      free(A);
      free(B);
      free(C);
    }

    short iterations() { return 1;}

    int n, dim;
    interval<T> *A, *B, *C, alpha, beta;
};
BENCH_F_F(BoostBench, Gemm, BoostBenchGemm)


//----------------------- Symm Bench -----------------------------------------//


template <class T>
class BoostBenchSymm : public BoostBench  {
   public:

    void begin() {
      n = r->length;
      dim = std::sqrt(n);
      alpha = interval<T>((r->alpha).inf(), (r->alpha).sup());
      beta = interval<T>((r->beta).inf(), (r->beta).sup());
      A = (interval<T>*) malloc(n*sizeof(interval<T>));
      B = (interval<T>*) malloc(n*sizeof(interval<T>));
      C = (interval<T>*) malloc(n*sizeof(interval<T>));
      acopy(n, r->x, A);
      acopy(n, r->y, B);
      acopy(n, r->y, C);
    }

    void run() {
      bi::symm('l', 'u', dim, dim, alpha, beta, A, B, C);
    }

    void end() {
      free(A);
      free(B);
      free(C);
    }

    short iterations() { return 1;}

    int n, dim;
    interval<T> *A, *B, *C, alpha, beta;
};
BENCH_F_F(BoostBench, Symm, BoostBenchSymm)


//----------------------- Syrk Bench -----------------------------------------//


template <class T>
class BoostBenchSyrk : public BoostBench  {
   public:

    void begin() {
      n = r->length;
      dim = std::sqrt(n);
      alpha = interval<T>((r->alpha).inf(), (r->alpha).sup());
      beta = interval<T>((r->beta).inf(), (r->beta).sup());
      A = (interval<T>*) malloc(n*sizeof(interval<T>));
      B = (interval<T>*) malloc(n*sizeof(interval<T>));
      acopy(n, r->x, A);
      acopy(n, r->y, B);
    }

    void run() {
      bi::syrk( 'u', dim, dim, alpha, beta, A, B);
    }

    void end() {
      free(A);
      free(B);
    }

    short iterations() { return 1;}

    int n, dim;
    interval<T> *A, *B, alpha, beta;
};
BENCH_F_F(BoostBench, Syrk, BoostBenchSyrk)


//----------------------- Syr2k Bench -----------------------------------------//


template <class T>
class BoostBenchSyr2k : public BoostBench  {
   public:

    void begin() {
      n = r->length;
      dim = std::sqrt(n);
      alpha = interval<T>((r->alpha).inf(), (r->alpha).sup());
      beta = interval<T>((r->beta).inf(), (r->beta).sup());
      A = (interval<T>*) malloc(n*sizeof(interval<T>));
      B = (interval<T>*) malloc(n*sizeof(interval<T>));
      C = (interval<T>*) malloc(n*sizeof(interval<T>));
      acopy(n, r->x, A);
      acopy(n, r->y, B);
      acopy(n, r->y, C);
    }

    void run() {
      bi::syr2k('u', dim, dim, alpha, beta, A, B, C);
    }

    void end() {
      free(A);
      free(B);
      free(C);
    }

    short iterations() { return 1;}

    int n, dim;
    interval<T> *A, *B, *C, alpha, beta;
};
BENCH_F_F(BoostBench, Syr2k, BoostBenchSyr2k)


//----------------------- Trmm Bench -----------------------------------------//


template <class T>
class BoostBenchTrmm : public BoostBench  {
   public:

    void begin() {
      n = r->length;
      dim = std::sqrt(n);
      alpha = interval<T>((r->alpha).inf(), (r->alpha).sup());
      A = (interval<T>*) malloc(n*sizeof(interval<T>));
      B = (interval<T>*) malloc(n*sizeof(interval<T>));
      acopy(n, r->x, A);
      acopy(n, r->y, B);
    }

    void run() {
      bi::trmm('l', 'u', dim, dim, alpha, A, B);
    }

    void end() {
      free(A);
      free(B);
    }

    short iterations() { return 1;}

    int n, dim;
    interval<T> *A, *B, alpha;
};
BENCH_F_F(BoostBench, Trmm, BoostBenchTrmm)


//----------------------- Trsm Bench -----------------------------------------//


template <class T>
class BoostBenchTrsm : public BoostBench  {
   public:

    void begin() {
      n = r->length;
      dim = std::sqrt(n);
      alpha = interval<T>((r->alpha).inf(), (r->alpha).sup());
      A = (interval<T>*) malloc(n*sizeof(interval<T>));
      B = (interval<T>*) malloc(n*sizeof(interval<T>));
      acopy(n, r->x, A);
      acopy(n, r->y, B);
      for(int i = 0; i < dim; ++i)
        A[i*dim+i] = abs(A[i*dim+i]) + interval<T>(0.1);
    }

    void run() {
      bi::trsm('l', 'u', dim, dim, alpha, A, B);
    }

    void end() {
      free(A);
      free(B);
    }

    short iterations() { return 1;}

    int n, dim;
    interval<T> *A, *B, alpha;
};
BENCH_F_F(BoostBench, Trsm, BoostBenchTrsm)



} // namespace bench
} // namespace intlag

#endif



