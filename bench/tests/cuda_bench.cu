#include "hip/hip_runtime.h"

//------------------------------------------------------------------------------
// Copyright (c) 2014 Ademar Marques Lacerda Filho
//
// This Source Code Form is subject to the terms of the Mozilla Public
// License, v. 2.0. If a copy of the MPL was not distributed with this
// file, You can obtain one at http://mozilla.org/MPL/2.0/.
//------------------------------------------------------------------------------


#ifndef CUDA_BENCH_H
#define CUDA_BENCH_H

//#include <cmath>

#include "aux/case.h"
#include "aux/reference.h"

//#include "../include/blas/omp_blas.h"
#include "../include/blas/cuda_blas.h"

#include "gtest/gtest.h"
#include "aux/test_interval.h"



namespace intlag {
namespace bench {


//----------------------- Fixture --------------------------------------------//
class CudaBench : public BenchTest {
  public:

    CudaBench() {
      r = Reference::getInstance();

      int n = r->length;
      CudaInterval<double> alpha = r->alpha;
      CudaInterval<double> *x = new CudaInterval<double> [n];
      acopy(n, r->x, x);
      scal(n, alpha, x);
    }
    virtual ~CudaBench() {}

    void SetUp() {}

    void TearDown() {}

    Reference* r;
};


//----------------------- Scal Bench -----------------------------------------//

template <class T>
class CudaBenchScal : public CudaBench  {
   public:

    void begin() {
      n = r->length;
      alpha = r->alpha;
      x = new CudaInterval<T> [n];
      acopy(n, r->x, x);
    }

    void run() {
      scal(n, alpha, x);
    }

    void end() {
      delete[] x;
    }

    int n;
    CudaInterval<T> *x, alpha;
};
BENCH_FD_F(CudaBench, Scal, CudaBenchScal)


//----------------------- AXPY Bench -----------------------------------------//

template <class T>
class CudaBenchAXPY : public CudaBench  {
   public:

    void begin() {
      n = r->length;
      alpha = r->alpha;
      x = new CudaInterval<T> [n];
      y = new CudaInterval<T> [n];
      acopy(n, r->x, x);
      acopy(n, r->y, y);
    }

    void run() {
      axpy(n, alpha, x, y);
    }

    void end() {
      delete[] x;
      delete[] y;
    }

    int n;
    CudaInterval<T> *x, *y, alpha;
};
BENCH_FD_F(CudaBench, AXPY, CudaBenchAXPY)


//----------------------- ASum Bench -----------------------------------------//

template <class T>
class CudaBenchASum : public CudaBench  {
   public:

    void begin() {
      n = r->length;
      x = new CudaInterval<T> [n];
      acopy(n, r->x, x);
    }

    void run() {
      asum(n, &ret, x);
    }

    void end() {
      delete[] x;
    }

    int n;
    CudaInterval<T> *x, ret;
};
BENCH_FD_F(CudaBench, Asum, CudaBenchASum)


//----------------------- Dot Bench -----------------------------------------//

template <class T>
class CudaBenchDot : public CudaBench  {
   public:

    void begin() {
      n = r->length;
      x = new CudaInterval<T> [n];
      y = new CudaInterval<T> [n];
      acopy(n, r->x, x);
      acopy(n, r->y, y);
    }

    void run() {
      dot(n, &ret, x, y);
    }

    void end() {
      delete[] x;
      delete[] y;
    }

    int n;
    CudaInterval<T> *x, *y, ret;
};
BENCH_FD_F(CudaBench, Dot, CudaBenchDot)


//----------------------- Norm2 Bench -----------------------------------------//

template <class T>
class CudaBenchNorm2 : public CudaBench  {
   public:

    void begin() {
      n = r->length;
      x = new CudaInterval<T> [n];
      acopy(n, r->x, x);
    }

    void run() {
      CudaGeneralManaged::norm2(n, &ret, x);
    }

    void end() {
      delete[] x;
    }

    int n;
    CudaInterval<T> *x, ret;
};
BENCH_FD_F(CudaBench, Norm2, CudaBenchNorm2)


//----------------------- Rot Bench -----------------------------------------//

template <class T>
class CudaBenchRot : public CudaBench  {
   public:

    void begin() {
      n = r->length;
      x = new CudaInterval<T> [n];
      y = new CudaInterval<T> [n];
      acopy(n, r->x, x);
      acopy(n, r->y, y);
    }

    void run() {
      CudaGeneralManaged::rot(n, x, y, 0.5, 0.5);
    }

    void end() {
      delete[] x;
      delete[] y;
    }

    int n;
    CudaInterval<T> *x, *y;
};
BENCH_FD_F(CudaBench, Rot, CudaBenchRot)


//----------------------- Rotm Bench -----------------------------------------//

template <class T>
class CudaBenchRotm : public CudaBench  {
   public:

    void begin() {
      h[0] = -1; h[1] = 0.5; h[2] = 0.6; h[3] = 0.7; h[4] = 0.8;
      n = r->length;
      x = new CudaInterval<T> [n];
      acopy(n, r->x, x);
      y = new CudaInterval<T> [n];
      acopy(n, r->y, y);
    }

    void run() {
      CudaGeneralManaged::rotm(n, x, y, h);
    }

    void end() {
      delete[] x;
      delete[] y;
    }

    int n;
    T h[5];
    CudaInterval<T> *x, *y;
};
BENCH_FD_F(CudaBench, Rotm, CudaBenchRotm)


//----------------------- Ger Bench -----------------------------------------//


template <class T>
class CudaBenchGer : public CudaBench  {
   public:

    void begin() {
      n = r->length;
      dim = std::sqrt(n);
      alpha = r->alpha;
      A = new CudaInterval<T> [n];
      x = new CudaInterval<T> [dim];
      y = new CudaInterval<T> [dim];
      acopy(n, r->x, A);
      acopy(dim, r->x, x);
      acopy(dim, r->y, y);
    }

    void run() {
      CudaGeneralManaged::ger(dim, dim, alpha, x, y, A);
    }

    void end() {
      delete[] A;
      delete[] x;
      delete[] y;
    }

    int n, dim;
    CudaInterval<T> *A, *x, *y, alpha, beta;
};
BENCH_FD_F(CudaBench, Ger, CudaBenchGer)


//----------------------- Gemv Bench -----------------------------------------//


template <class T>
class CudaBenchGemv : public CudaBench  {
   public:

    void begin() {
      n = r->length;
      dim = std::sqrt(n);
      alpha = r->alpha;
      beta = r->beta;
      A = new CudaInterval<T> [n];
      x = new CudaInterval<T> [dim];
      y = new CudaInterval<T> [dim];
      acopy(n, r->x, A);
      acopy(dim, r->y, x);
      acopy(dim, r->y, y);
    }

    void run() {
      //DeviceData< CudaInterval<T> > dA(n, A), dx(dim, x), dy(dim, y);
      //CudaGeneralManaged::gemv(dim, dim, alpha, beta, dA.data(), dx.data(), dy.data());
      CudaGeneralManaged::gemv(dim, dim, alpha, beta, A, x, y);
      //dy.toHost(y);
    }

    void end() {
      delete[] A;
      delete[] x;
      delete[] y;
    }

    int n, dim;
    CudaInterval<T> *A, *x, *y, alpha, beta;
};
BENCH_FD_F(CudaBench, Gemv, CudaBenchGemv)


//----------------------- Gemv Bench -----------------------------------------//


template <class T>
class CudaBenchGemvShared : public CudaBench  {
   public:

    void begin() {
      n = r->length;
      dim = std::sqrt(n);
      alpha = r->alpha;
      beta = r->beta;
      A = new CudaInterval<T> [n];
      x = new CudaInterval<T> [dim];
      y = new CudaInterval<T> [dim];
      acopy(n, r->x, A);
      acopy(dim, r->y, x);
      acopy(dim, r->y, y);
    }

    void run() {
      //DeviceData< CudaInterval<T> > dA(n, A), dx(dim, x), dy(dim, y);
      //CudaGeneralManaged::gemv(dim, dim, alpha, beta, dA.data(), dx.data(), dy.data());
      CudaSharedManaged::gemv(dim, dim, alpha, beta, A, x, y);
      //dy.toHost(y);
    }

    void end() {
      delete[] A;
      delete[] x;
      delete[] y;
    }

    int n, dim;
    CudaInterval<T> *A, *x, *y, alpha, beta;
};
BENCH_FD_F(CudaBench, GemvShared, CudaBenchGemvShared)


//----------------------- Gemm Bench -----------------------------------------//

template <class T>
class CudaBenchGemm : public CudaBench  {
   public:

    void begin() {
      n = r->length;
      dim = std::sqrt(n);
      alpha = r->alpha;
      beta = r->beta;
      A = new CudaInterval<T> [n];
      B = new CudaInterval<T> [n];
      C = new CudaInterval<T> [n];
      acopy(n, r->x, A);
      acopy(n, r->y, B);
      acopy(n, r->y, C);
    }

    void run() {
      //DeviceData< CudaInterval<T> > dA(n, A);
      //DeviceData< CudaInterval<T> > dB(n, B);
      //DeviceData< CudaInterval<T> > dC(n, C);
      //gemm(dim, dim, dim, alpha, beta, dA.data(), dB.data(), dC.data());
      CudaGeneralManaged::gemm(dim, dim, dim, alpha, beta, A, B, C);
      //dC.toHost(C);
    }

    void check() {
      CudaInterval<T> *x, *y, *z;
      x = (CudaInterval<T>*) malloc(n*sizeof(CudaInterval<T>));
      y = (CudaInterval<T>*) malloc(n*sizeof(CudaInterval<T>));
      z = (CudaInterval<T>*) malloc(n*sizeof(CudaInterval<T>));
      acopy(n, r->x, x);
      acopy(n, r->y, y);
      acopy(n, r->y, z);
      acopy(n, r->y, C);

      gemm(dim, dim, dim, alpha, beta, x, y, z);
      CudaGeneralManaged::gemm(dim, dim, dim, alpha, beta, A, B, C);

      for(int i = 0; i < n; ++i)
        EXPECT_DINTERVAL_NEAR(z[i], C[i], 0.1);

      free(x);
      free(y);
      free(z);
    }

    short iterations() { return 100;}

    void end() {
      delete[] A;
      delete[] B;
      delete[] C;
    }


    int n, dim;
    CudaInterval<T> *A, *B, *C, alpha, beta;
};
BENCH_FD_F(CudaBench, Gemm, CudaBenchGemm)


//----------------------- GemmShared Bench -----------------------------------------//


template <class T>
class CudaBenchGemmShared : public CudaBench  {
   public:

    void begin() {
      n = r->length;
      dim = std::sqrt(n);
      alpha = r->alpha;
      beta = r->beta;
      A = new CudaInterval<T> [n];
      B = new CudaInterval<T> [n];
      C = new CudaInterval<T> [n];
      acopy(n, r->x, A);
      acopy(n, r->y, B);
      acopy(n, r->y, C);
    }

    void run() {
      //DeviceData< CudaInterval<T> > dA(n, A);
      //DeviceData< CudaInterval<T> > dB(n, B);
      //DeviceData< CudaInterval<T> > dC(n, C);
      //gemm(dim, dim, dim, alpha, beta, dA.data(), dB.data(), dC.data());
      CudaSharedManaged::gemm(dim, dim, dim, alpha, beta, A, B, C);
      //dC.toHost(C);
    }

    void check() {
      CudaInterval<T> *x, *y, *z;
      x = (CudaInterval<T>*) malloc(n*sizeof(CudaInterval<T>));
      y = (CudaInterval<T>*) malloc(n*sizeof(CudaInterval<T>));
      z = (CudaInterval<T>*) malloc(n*sizeof(CudaInterval<T>));
      acopy(n, r->x, x);
      acopy(n, r->y, y);
      acopy(n, r->y, z);
      acopy(n, r->y, C);

      gemm(dim, dim, dim, alpha, beta, x, y, z);
      CudaSharedManaged::gemm(dim, dim, dim, alpha, beta, A, B, C);

      for(int i = 0; i < n; ++i)
        EXPECT_DINTERVAL_NEAR(z[i], C[i], 0.1);

      free(x);
      free(y);
      free(z);
    }

    short iterations() { return 100;}

    void end() {
      delete[] A;
      delete[] B;
      delete[] C;
    }


    int n, dim;
    CudaInterval<T> *A, *B, *C, alpha, beta;
};
BENCH_FD_F(CudaBench, GemmShared, CudaBenchGemmShared)


} // namespace bench
} // namespace intlag

#endif



