#include "hip/hip_runtime.h"

//------------------------------------------------------------------------------
// Copyright (c) 2014 Ademar Marques Lacerda Filho
//
// This Source Code Form is subject to the terms of the Mozilla Public
// License, v. 2.0. If a copy of the MPL was not distributed with this
// file, You can obtain one at http://mozilla.org/MPL/2.0/.
//------------------------------------------------------------------------------


#ifndef CUDA_BENCH_KERNEL_H
#define CUDA_BENCH_KERNEL_H

//#include <cmath>

#include "aux/case.h"
#include "aux/reference.h"

//#include "../include/blas/omp_blas.h"
#include "../include/blas/cuda_blas.h"

#include "gtest/gtest.h"
#include "aux/test_interval.h"



namespace intlag {
namespace bench {


//----------------------- Fixture --------------------------------------------//
class CudaKernelBench : public BenchTest {
  public:

    CudaKernelBench() {
      r = Reference::getInstance();
    }
    virtual ~CudaKernelBench() {}

    void SetUp() {
      int n = r->length;
      CudaInterval<double> alpha = r->alpha;
      CudaInterval<double> *x = new CudaInterval<double> [n];
      acopy(n, r->x, x);
      scal(n, alpha, x);
    }

    void TearDown() {}

    Reference* r;
};


//----------------------- Scal Bench -----------------------------------------//

template <class T>
class CudaKernelBenchScal : public CudaKernelBench  {
   public:

    void begin() {
      n = r->length;
      alpha = r->alpha;
      x = new CudaInterval<T> [n];
      acopy(n, r->x, x);
    }

    void run() {
      DeviceData< CudaInterval<T> > dx(n,x);
	    //CudaGeneral::scal(n, alpha, dx.data());
      dx.toHost(x);
    }

    void end() {
      delete[] x;
    }

    int n;
    CudaInterval<T> *x, alpha;
};
//BENCH_FD_F(CudaKernelBench, Scal, CudaKernelBenchScal)


//----------------------- AXPY Bench -----------------------------------------//

template <class T>
class CudaKernelBenchAXPY : public CudaKernelBench  {
   public:

    void begin() {
      n = r->length;
      alpha = r->alpha;
      x = new CudaInterval<T> [n];
      y = new CudaInterval<T> [n];
      acopy(n, r->x, x);
      acopy(n, r->y, y);
    }

    void run() {
      DeviceData< CudaInterval<T> > dx(n,x), dy(n,y);
      //CudaGeneral::axpy(n, alpha, dx.data(), dy.data());
      dy.toHost(y);
    }

    void end() {
      delete[] x;
      delete[] y;
    }

    int n;
    CudaInterval<T> *x, *y, alpha;
};
BENCH_FD_F(CudaKernelBench, AXPY, CudaKernelBenchAXPY)

/*
//----------------------- ASum Bench -----------------------------------------//

template <class T>
class CudaKernelBenchASum : public CudaKernelBench  {
   public:

    void begin() {
      n = r->length;
      x = new CudaInterval<T> [n];
      acopy(n, r->x, x);
    }

    void run() {
      CudaGeneral::asum(n, &ret, x);
    }

    void end() {
      delete[] x;
    }

    int n;
    CudaInterval<T> *x, ret;
};
BENCH_F_F(CudaKernelBench, Asum, CudaKernelBenchASum)


//----------------------- Dot Bench -----------------------------------------//

template <class T>
class CudaKernelBenchDot : public CudaKernelBench  {
   public:

    void begin() {
      n = r->length;
      x = new CudaInterval<T> [n];
      y = new CudaInterval<T> [n];
      acopy(n, r->x, x);
      acopy(n, r->y, y);
    }

    void run() {
      CudaGeneral::dot(n, &ret, x, y);
    }

    void end() {
      delete[] x;
      delete[] y;
    }

    int n;
    CudaInterval<T> *x, *y, ret;
};
BENCH_F_F(CudaKernelBench, Dot, CudaKernelBenchDot)


//----------------------- Norm2 Bench -----------------------------------------//

template <class T>
class CudaKernelBenchNorm2 : public CudaKernelBench  {
   public:

    void begin() {
      n = r->length;
      x = new CudaInterval<T> [n];
      acopy(n, r->x, x);
    }

    void run() {
      CudaGeneral::norm2(n, &ret, x);
    }

    void end() {
      delete[] x;
    }

    int n;
    CudaInterval<T> *x, ret;
};
BENCH_F_F(CudaKernelBench, Norm2, CudaKernelBenchNorm2)

//----------------------- Rot Bench -----------------------------------------//

template <class T>
class CudaKernelBenchRot : public CudaKernelBench  {
   public:

    void begin() {
      n = r->length;
      x = new CudaInterval<T> [n];
      y = new CudaInterval<T> [n];
      acopy(n, r->x, x);
      acopy(n, r->y, y);
    }

    void run() {
      CudaGeneral::rot(n, x, y, 0.5, 0.5);
    }

    void end() {
      delete[] x;
      delete[] y;
    }

    int n;
    CudaInterval<T> *x, *y;
};
BENCH_F_F(CudaKernelBench, Rot, CudaKernelBenchRot)


//----------------------- Rotm Bench -----------------------------------------//

template <class T>
class CudaKernelBenchRotm : public CudaKernelBench  {
   public:

    void begin() {
      h[0] = -1; h[1] = 0.5; h[2] = 0.6; h[3] = 0.7; h[4] = 0.8;
      n = r->length;
      x = new CudaInterval<T> [n];
      acopy(n, r->x, x);
      y = new CudaInterval<T> [n];
      acopy(n, r->y, y);
    }

    void run() {
      CudaGeneral::rotm(n, x, y, h);
    }

    void end() {
      delete[] x;
      delete[] y;
    }

    int n;
    T h[5];
    CudaInterval<T> *x, *y;
};
BENCH_F_F(CudaKernelBench, Rotm, CudaKernelBenchRotm)


//----------------------- Ger Bench -----------------------------------------//


template <class T>
class CudaKernelBenchGer : public CudaKernelBench  {
   public:

    void begin() {
      n = r->length;
      dim = std::sqrt(n);
      alpha = r->alpha;
      A = new CudaInterval<T> [n];
      x = new CudaInterval<T> [dim];
      y = new CudaInterval<T> [dim];
      acopy(n, r->x, A);
      acopy(dim, r->x, x);
      acopy(dim, r->y, y);
    }

    void run() {
      CudaGeneral::ger(dim, dim, alpha, x, y, A);
    }

    void end() {
      delete[] A;
      delete[] x;
      delete[] y;
    }

    int n, dim;
    CudaInterval<T> *A, *x, *y, alpha, beta;
};
BENCH_F_F(CudaKernelBench, Ger, CudaKernelBenchGer)
*/

//----------------------- Gemv Bench -----------------------------------------//


template <class T>
class CudaKernelBenchGemv : public CudaKernelBench  {
   public:

    void begin() {
      n = r->length;
      dim = std::sqrt(n);
      alpha = r->alpha;
      beta = r->beta;
      A = new CudaInterval<T> [n];
      x = new CudaInterval<T> [dim];
      y = new CudaInterval<T> [dim];
      acopy(n, r->x, A);
      acopy(dim, r->y, x);
      acopy(dim, r->y, y);
    }

    void run() {
      DeviceData< CudaInterval<T> > dA(n, A), dx(dim, x), dy(dim, y);
      //CudaGeneralManaged::gemv(dim, dim, alpha, beta, dA.data(), dx.data(), dy.data());
      //CudaGeneral::gemv(dim, dim, alpha, beta, A, x, y);
      dy.toHost(y);
    }

    void end() {
      delete[] A;
      delete[] x;
      delete[] y;
    }

    int n, dim;
    CudaInterval<T> *A, *x, *y, alpha, beta;
};
BENCH_FD_F(CudaKernelBench, Gemv, CudaKernelBenchGemv)


//----------------------- Gemv Bench -----------------------------------------//

/*
template <class T>
class CudaKernelBenchGemvShared : public CudaKernelBench  {
   public:

    void begin() {
      n = r->length;
      dim = std::sqrt(n);
      alpha = r->alpha;
      beta = r->beta;
      A = new CudaInterval<T> [n];
      x = new CudaInterval<T> [dim];
      y = new CudaInterval<T> [dim];
      acopy(n, r->x, A);
      acopy(dim, r->y, x);
      acopy(dim, r->y, y);
    }

    void run() {
      //DeviceData< CudaInterval<T> > dA(n, A), dx(dim, x), dy(dim, y);
      //CudaGeneralManaged::gemv(dim, dim, alpha, beta, dA.data(), dx.data(), dy.data());
      CudaShared::gemv(dim, dim, alpha, beta, A, x, y);
      //dy.toHost(y);
    }

    void end() {
      delete[] A;
      delete[] x;
      delete[] y;
    }

    int n, dim;
    CudaInterval<T> *A, *x, *y, alpha, beta;
};
BENCH_FD_F(CudaKernelBench, GemvShared, CudaKernelBenchGemvShared)

*/
//----------------------- Gemm Bench -----------------------------------------//

template <class T>
class CudaKernelBenchGemm : public CudaKernelBench  {
   public:

    void begin() {
      n = r->length;
      dim = std::sqrt(n);
      alpha = r->alpha;
      beta = r->beta;
      A = new CudaInterval<T> [n];
      B = new CudaInterval<T> [n];
      C = new CudaInterval<T> [n];
      acopy(n, r->x, A);
      acopy(n, r->y, B);
      acopy(n, r->y, C);
    }

    void run() {
      DeviceData< CudaInterval<T> > dA(n, A);
      DeviceData< CudaInterval<T> > dB(n, B);
      DeviceData< CudaInterval<T> > dC(n, C);
      //gemm(dim, dim, dim, alpha, beta, dA.data(), dB.data(), dC.data());
      //CudaGeneral::gemm(dim, dim, dim, alpha, beta, dx.data(), dy.data(), dz.data());
      dC.toHost(C);
    }

    void check() {
    }

    short iterations() { return 100;}

    void end() {
      delete[] A;
      delete[] B;
      delete[] C;
    }


    int n, dim;
    CudaInterval<T> *A, *B, *C, alpha, beta;
};
BENCH_FD_F(CudaKernelBench, Gemm, CudaKernelBenchGemm)
/*

//----------------------- GemmShared Bench -----------------------------------------//


template <class T>
class CudaKernelBenchGemmShared : public CudaKernelBench  {
   public:

    void begin() {
      n = r->length;
      dim = std::sqrt(n);
      alpha = r->alpha;
      beta = r->beta;
      A = new CudaInterval<T> [n];
      B = new CudaInterval<T> [n];
      C = new CudaInterval<T> [n];
      acopy(n, r->x, A);
      acopy(n, r->y, B);
      acopy(n, r->y, C);
    }

    void run() {
      //DeviceData< CudaInterval<T> > dA(n, A);
      //DeviceData< CudaInterval<T> > dB(n, B);
      //DeviceData< CudaInterval<T> > dC(n, C);
      //gemm(dim, dim, dim, alpha, beta, dA.data(), dB.data(), dC.data());
      CudaShared::gemm(dim, dim, dim, alpha, beta, A, B, C);
      //dC.toHost(C);
    }

    void check() {
      CudaInterval<T> *x, *y, *z;
      x = (CudaInterval<T>*) malloc(n*sizeof(CudaInterval<T>));
      y = (CudaInterval<T>*) malloc(n*sizeof(CudaInterval<T>));
      z = (CudaInterval<T>*) malloc(n*sizeof(CudaInterval<T>));
      acopy(n, r->x, x);
      acopy(n, r->y, y);
      acopy(n, r->y, z);
      acopy(n, r->y, C);

      gemm(dim, dim, dim, alpha, beta, x, y, z);
      CudaShared::gemm(dim, dim, dim, alpha, beta, A, B, C);

      for(int i = 0; i < n; ++i)
        EXPECT_DINTERVAL_NEAR(z[i], C[i], 0.1);

      free(x);
      free(y);
      free(z);
    }

    short iterations() { return 1;}

    void end() {
      delete[] A;
      delete[] B;
      delete[] C;
    }


    int n, dim;
    CudaInterval<T> *A, *B, *C, alpha, beta;
};
BENCH_FD_F(CudaKernelBench, GemmShared, CudaKernelBenchGemmShared)*/


} // namespace bench
} // namespace intlag

#endif



