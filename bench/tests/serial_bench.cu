#include "hip/hip_runtime.h"

//------------------------------------------------------------------------------
// Copyright (c) 2014 Ademar Marques Lacerda Filho
//
// This Source Code Form is subject to the terms of the Mozilla Public
// License, v. 2.0. If a copy of the MPL was not distributed with this
// file, You can obtain one at http://mozilla.org/MPL/2.0/.
//------------------------------------------------------------------------------


#ifndef SERIAL_BENCH_H
#define SERIAL_BENCH_H


#include "aux/case.h"
#include "aux/reference.h"

#include "../include/blas/serial_blas.h"

#include "gtest/gtest.h"
#include "aux/test_interval.h"


namespace intlag {
namespace bench {


//----------------------- Fixture --------------------------------------------//

class SerialBench : public BenchTest {
  public:

    SerialBench() {
      r = Reference::getInstance();
    }
    virtual ~SerialBench() {}

    void SetUp() {}
    void TearDown() {}

    Reference* r;
};


//----------------------- Scal Bench -----------------------------------------//

template <class T>
class SerialBenchScal : public SerialBench  {
   public:

    void begin() {
      n = r->length;
      alpha = r->alpha;
      x = new Interval<T> [n];
      acopy(n, r->x, x);
    }

    void run() {
      scal(n, alpha, x);
    }

    void end() {
      delete[] x;
    }

    int n;
    Interval<T> *x, alpha;
};
BENCH_FD_F(SerialBench, Scal, SerialBenchScal)


//----------------------- AXPY Bench -----------------------------------------//

template <class T>
class SerialBenchAXPY : public SerialBench  {
   public:

    void begin() {
      n = r->length;
      alpha = r->alpha;
      x = new Interval<T> [n];
      y = new Interval<T> [n];
      acopy(n, r->x, x);
      acopy(n, r->y, y);
    }

    void run() {
      axpy(n, alpha, x, y);
    }

    void end() {
      delete[] x;
      delete[] y;
    }

    int n;
    Interval<T> *x, *y, alpha;
};
BENCH_FD_F(SerialBench, AXPY, SerialBenchAXPY)


//----------------------- ASum Bench -----------------------------------------//

template <class T>
class SerialBenchASum : public SerialBench  {
   public:

    void begin() {
      n = r->length;
      x = new Interval<T> [n];
      acopy(n, r->x, x);
    }

    void run() {
      asum(n, &ret, x);
    }

    void end() {
      delete[] x;
    }

    int n;
    Interval<T> *x, ret;
};
BENCH_F_F(SerialBench, Asum, SerialBenchASum)


//----------------------- Dot Bench -----------------------------------------//

template <class T>
class SerialBenchDot : public SerialBench  {
   public:

    void begin() {
      n = r->length;
      x = new Interval<T> [n];
      y = new Interval<T> [n];
      acopy(n, r->x, x);
      acopy(n, r->y, y);
    }

    void run() {
      dot(n, &ret, x, y);
    }

    void end() {
      delete[] x;
      delete[] y;
    }

    int n;
    Interval<T> *x, *y, ret;
};
BENCH_F_F(SerialBench, Dot, SerialBenchDot)


//----------------------- Norm2 Bench -----------------------------------------//

template <class T>
class SerialBenchNorm2 : public SerialBench  {
   public:

    void begin() {
      n = r->length;
      x = new Interval<T> [n];
      acopy(n, r->x, x);
    }

    void run() {
      norm2(n, &ret, x);
    }

    void end() {
      delete[] x;
    }

    int n;
    Interval<T> *x, ret;
};
BENCH_F_F(SerialBench, Norm2, SerialBenchNorm2)


//----------------------- Rot Bench -----------------------------------------//

template <class T>
class SerialBenchRot : public SerialBench  {
   public:

    void begin() {
      n = r->length;
      x = new Interval<T> [n];
      acopy(n, r->x, x);
      y = new Interval<T> [n];
      acopy(n, r->y, y);
    }

    void run() {
      rot(n, x, y, 0.5, 0.5);
    }

    void end() {
      delete[] x;
      delete[] y;
    }

    int n;
    Interval<T> *x, *y;
};
BENCH_F_F(SerialBench, Rot, SerialBenchRot)


//----------------------- Rotm Bench -----------------------------------------//

template <class T>
class SerialBenchRotm : public SerialBench  {
   public:

    void begin() {
      h[0] = -1; h[1] = 0.5; h[2] = 0.6; h[3] = 0.7; h[4] = 0.8;
      n = r->length;
      x = new Interval<T> [n];
      acopy(n, r->x, x);
      y = new Interval<T> [n];
      acopy(n, r->y, y);
    }

    void run() {
      rotm(n, x, y, h);
    }

    void end() {
      delete[] x;
      delete[] y;
    }

    int n;
    T h[5];
    Interval<T> *x, *y;
};
BENCH_F_F(SerialBench, Rotm, SerialBenchRotm)


//----------------------- Ger Bench -----------------------------------------//


template <class T>
class SerialBenchGer : public SerialBench  {
   public:

    void begin() {
      n = r->length;
      dim = std::sqrt(n);
      alpha = r->alpha;
      A = new Interval<T> [n];
      x = new Interval<T> [dim];
      y = new Interval<T> [dim];
      acopy(n, r->x, A);
      acopy(dim, r->x, x);
      acopy(dim, r->y, y);
    }

    void run() {
      ger(dim, dim, alpha, x, y, A);
    }

    void end() {
      delete[] A;
      delete[] x;
      delete[] y;
    }

    int n, dim;
    Interval<T> *A, *x, *y, alpha, beta;
};
BENCH_F_F(SerialBench, Ger, SerialBenchGer)


//----------------------- Syr Bench -----------------------------------------//


template <class T>
class SerialBenchSyr : public SerialBench  {
   public:

    void begin() {
      n = r->length;
      dim = std::sqrt(n);
      alpha = r->alpha;
      A = new Interval<T> [n];
      x = new Interval<T> [dim];
      acopy(n, r->x, A);
      acopy(dim, r->x, x);
    }

    void run() {
      syr('u', dim, alpha, x, A);
    }

    void end() {
      delete[] A;
      delete[] x;
    }

    int n, dim;
    Interval<T> *A, *x, alpha, beta;
};
BENCH_F_F(SerialBench, Syr, SerialBenchSyr)


//----------------------- Syr2 Bench -----------------------------------------//


template <class T>
class SerialBenchSyr2 : public SerialBench  {
   public:

    void begin() {
      n = r->length;
      dim = std::sqrt(n);
      alpha = r->alpha;
      A = new Interval<T> [n];
      x = new Interval<T> [dim];
      y = new Interval<T> [dim];
      acopy(n, r->x, A);
      acopy(dim, r->x, x);
      acopy(dim, r->y, y);
    }

    void run() {
      syr2('u', dim, alpha, x, y, A);
    }

    void end() {
      delete[] A;
      delete[] x;
      delete[] y;
    }

    int n, dim;
    Interval<T> *A, *x, *y, alpha, beta;
};
BENCH_F_F(SerialBench, Syr2, SerialBenchSyr2)


//----------------------- Spr Bench -----------------------------------------//


template <class T>
class SerialBenchSpr : public SerialBench  {
   public:

    void begin() {
      n = r->length;
      dim = std::sqrt(n);
      alpha = r->alpha;
      A = new Interval<T> [n];
      x = new Interval<T> [dim];
      acopy(n, r->x, A);
      acopy(dim, r->x, x);
    }

    void run() {
      spr('u', dim, alpha, x, A);
    }

    void end() {
      delete[] A;
      delete[] x;
    }

    int n, dim;
    Interval<T> *A, *x, alpha, beta;
};
BENCH_F_F(SerialBench, Spr, SerialBenchSpr)


//----------------------- Spr2 Bench -----------------------------------------//


template <class T>
class SerialBenchSpr2 : public SerialBench  {
   public:

    void begin() {
      n = r->length;
      dim = std::sqrt(n);
      alpha = r->alpha;
      A = new Interval<T> [n];
      x = new Interval<T> [dim];
      y = new Interval<T> [dim];
      acopy(n, r->x, A);
      acopy(dim, r->x, x);
      acopy(dim, r->y, y);
    }

    void run() {
      spr2('u', dim, alpha, x, y, A);
    }

    void end() {
      delete[] A;
      delete[] x;
      delete[] y;
    }

    int n, dim;
    Interval<T> *A, *x, *y, alpha, beta;
};
BENCH_F_F(SerialBench, Spr2, SerialBenchSpr2)


//----------------------- Gemv Bench -----------------------------------------//


template <class T>
class SerialBenchGemv : public SerialBench  {
   public:

    void begin() {
      n = r->length;
      dim = std::sqrt(n);
      alpha = r->alpha;
      beta = r->beta;
      A = new Interval<T> [n];
      x = new Interval<T> [dim];
      y = new Interval<T> [dim];
      acopy(n, r->x, A);
      acopy(dim, r->x, x);
      acopy(dim, r->y, y);
    }

    void run() {
      gemv(dim, dim, alpha, beta, A, x, y);
    }

    void end() {
      delete[] A;
      delete[] x;
      delete[] y;
    }

    int n, dim;
    Interval<T> *A, *x, *y, alpha, beta;
};
BENCH_F_F(SerialBench, Gemv, SerialBenchGemv)


//----------------------- Gbmv Bench -----------------------------------------//


template <class T>
class SerialBenchGbmv : public SerialBench  {
   public:

    void begin() {
      n = r->length;
      dim = std::sqrt(n);
      band = dim/4;
      alpha = r->alpha;
      beta = r->beta;
      A = new Interval<T> [n];
      x = new Interval<T> [dim];
      y = new Interval<T> [dim];
      acopy(n, r->x, A);
      acopy(dim, r->x, x);
      acopy(dim, r->y, y);
    }

    void run() {
      gbmv(dim, dim, band, band, alpha, beta, A, x, y);
    }

    void end() {
      delete[] A;
      delete[] x;
      delete[] y;
    }

    int n, dim, band;
    Interval<T> *A, *x, *y, alpha, beta;
};
BENCH_F_F(SerialBench, Gbmv, SerialBenchGbmv)


//----------------------- Symv Bench -----------------------------------------//


template <class T>
class SerialBenchSymv : public SerialBench  {
   public:

    void begin() {
      n = r->length;
      dim = std::sqrt(n);
      alpha = r->alpha;
      beta = r->beta;
      A = new Interval<T> [n];
      x = new Interval<T> [dim];
      y = new Interval<T> [dim];
      acopy(n, r->x, A);
      acopy(dim, r->x, x);
      acopy(dim, r->y, y);
    }

    void run() {
      symv('u', dim, alpha, beta, A, x, y);
    }

    void end() {
      delete[] A;
      delete[] x;
      delete[] y;
    }

    int n, dim;
    Interval<T> *A, *x, *y, alpha, beta;
};
BENCH_F_F(SerialBench, Symv, SerialBenchSymv)


//----------------------- Sbmv Bench -----------------------------------------//


template <class T>
class SerialBenchSbmv : public SerialBench  {
   public:

    void begin() {
      n = r->length;
      dim = std::sqrt(n);
      band = dim/4;
      alpha = r->alpha;
      beta = r->beta;
      A = new Interval<T> [n];
      x = new Interval<T> [dim];
      y = new Interval<T> [dim];
      acopy(n, r->x, A);
      acopy(dim, r->x, x);
      acopy(dim, r->y, y);
    }

    void run() {
      sbmv('u', dim, band, alpha, beta, A, x, y);
    }

    void end() {
      delete[] A;
      delete[] x;
      delete[] y;
    }

    int n, dim, band;
    Interval<T> *A, *x, *y, alpha, beta;
};
BENCH_F_F(SerialBench, Sbmv, SerialBenchSbmv)


//----------------------- Spmv Bench -----------------------------------------//


template <class T>
class SerialBenchSpmv : public SerialBench  {
   public:

    void begin() {
      n = r->length;
      dim = std::sqrt(n);
      alpha = r->alpha;
      beta = r->beta;
      A = new Interval<T> [n];
      x = new Interval<T> [dim];
      y = new Interval<T> [dim];
      acopy(n, r->x, A);
      acopy(dim, r->x, x);
      acopy(dim, r->y, y);
    }

    void run() {
      spmv('u', dim, alpha, beta, A, x, y);
    }

    void end() {
      delete[] A;
      delete[] x;
      delete[] y;
    }

    int n, dim;
    Interval<T> *A, *x, *y, alpha, beta;
};
BENCH_F_F(SerialBench, Spmv, SerialBenchSpmv)


//----------------------- Trmv Bench -----------------------------------------//


template <class T>
class SerialBenchTrmv : public SerialBench  {
   public:

    void begin() {
      n = r->length;
      dim = std::sqrt(n);
      A = new Interval<T> [n];
      x = new Interval<T> [dim];
      acopy(n, r->y, A);
      acopy(dim, r->x, x);
    }

    void run() {
      trmv('u', dim, A, x);
    }

    void end() {
      delete[] A;
      delete[] x;
    }

    int n, dim;
    Interval<T> *A, *x;
};
BENCH_F_F(SerialBench, Trmv, SerialBenchTrmv)


//----------------------- Tbmv Bench -----------------------------------------//


template <class T>
class SerialBenchTbmv : public SerialBench  {
   public:

    void begin() {
      n = r->length;
      dim = std::sqrt(n);
      band = dim/4;
      A = new Interval<T> [n];
      x = new Interval<T> [dim];
      acopy(n, r->y, A);
      acopy(dim, r->x, x);
    }

    void run() {
      tbmv('u', dim, band, A, x);
    }

    void end() {
      delete[] A;
      delete[] x;
    }

    int n, dim, band;
    Interval<T> *A, *x;
};
BENCH_F_F(SerialBench, Tbmv, SerialBenchTbmv)


//----------------------- Tpmv Bench -----------------------------------------//


template <class T>
class SerialBenchTpmv : public SerialBench  {
   public:

    void begin() {
      n = r->length;
      dim = std::sqrt(n);
      A = new Interval<T> [n];
      x = new Interval<T> [dim];
      acopy(n, r->y, A);
      acopy(dim, r->x, x);
    }

    void run() {
      tpmv('u', dim, A, x);
    }

    void end() {
      delete[] A;
      delete[] x;
    }

    int n, dim;
    Interval<T> *A, *x;
};
BENCH_F_F(SerialBench, Tpmv, SerialBenchTpmv)


//----------------------- Trsv Bench -----------------------------------------//


template <class T>
class SerialBenchTrsv : public SerialBench  {
   public:

    void begin() {
      n = r->length;
      dim = std::sqrt(n);
      A = new Interval<T> [n];
      x = new Interval<T> [dim];
      acopy(n, r->y, A);
      acopy(dim, r->x, x);
      for(int i = 0; i < dim; ++i)
        A[i*dim+i] = intlag::abs(A[i*dim+i]) + 0.1;
    }

    void run() {
      trsv('u', dim, A, x);
    }

    void end() {

      for(int i = 0; i < dim; ++i)
        EXPECT_FALSE(isnan(x[i].inf()));

      delete[] A;
      delete[] x;
    }

    int n, dim;
    Interval<T> *A, *x;
};
BENCH_F_F(SerialBench, Trsv, SerialBenchTrsv)


//----------------------- Tbsv Bench -----------------------------------------//


template <class T>
class SerialBenchTbsv : public SerialBench  {
   public:

    void begin() {
      n = r->length;
      dim = std::sqrt(n);
      band = dim/4;
      A = new Interval<T> [n];
      x = new Interval<T> [dim];
      acopy(n, r->y, A);
      acopy(dim, r->x, x);
      for(int i = 0; i < dim; ++i)
        A[i*band+i] = intlag::abs(A[i*band+i]) + 0.1;
    }

    void run() {
      tbsv('u', dim, band, A, x);
    }

    void end() {

      for(int i = 0; i < dim; ++i)
        EXPECT_FALSE(isnan(x[i].inf()));

      delete[] A;
      delete[] x;
    }

    int n, dim, band;
    Interval<T> *A, *x;
};
BENCH_F_F(SerialBench, Tbsv, SerialBenchTbsv)


//----------------------- Tpsv Bench -----------------------------------------//


template <class T>
class SerialBenchTpsv : public SerialBench  {
   public:

    void begin() {
      n = r->length;
      dim = std::sqrt(n);
      A = new Interval<T> [n];
      x = new Interval<T> [dim];
      acopy(n, r->y, A);
      acopy(dim, r->x, x);
      for(int i = 0; i < dim; ++i)
        A[INDEX_TRIAN_UP(dim, i, i)] = intlag::abs(A[INDEX_TRIAN_UP(dim, i, i)]) + 0.1;
    }

    void run() {
      tpsv('u', dim, A, x);
    }

    void end() {

      for(int i = 0; i < dim; ++i)
        EXPECT_FALSE(isnan(x[i].inf()));

      delete[] A;
      delete[] x;
    }

    int n, dim;
    Interval<T> *A, *x;
};
BENCH_F_F(SerialBench, Tpsv, SerialBenchTpsv)


//----------------------- Gemm Bench -----------------------------------------//


template <class T>
class SerialBenchGemm : public SerialBench  {
   public:

    void begin() {
      n = r->length;
      dim = std::sqrt(n);
      alpha = r->alpha;
      beta = r->beta;
      A = (Interval<T>*) malloc(n*sizeof(Interval<T>));
      B = (Interval<T>*) malloc(n*sizeof(Interval<T>));
      C = (Interval<T>*) malloc(n*sizeof(Interval<T>));
      acopy(n, r->x, A);
      acopy(n, r->y, B);
      acopy(n, r->y, C);
    }

    void run() {
      gemm(dim, dim, dim, alpha, beta, A, B, C);
    }

    void end() {
      free(A);
      free(B);
      free(C);
    }

    short iterations() { return 1;}

    int n, dim;
    Interval<T> *A, *B, *C, alpha, beta;
};
BENCH_F_F(SerialBench, Gemm, SerialBenchGemm)


//----------------------- Symm Bench -----------------------------------------//


template <class T>
class SerialBenchSymm : public SerialBench  {
   public:

    void begin() {
      n = r->length;
      dim = std::sqrt(n);
      alpha = r->alpha;
      beta = r->beta;
      A = (Interval<T>*) malloc(n*sizeof(Interval<T>));
      B = (Interval<T>*) malloc(n*sizeof(Interval<T>));
      C = (Interval<T>*) malloc(n*sizeof(Interval<T>));
      acopy(n, r->x, A);
      acopy(n, r->y, B);
      acopy(n, r->y, C);
    }

    void run() {
      symm('l', 'u', dim, dim, alpha, beta, A, B, C);
    }

    void end() {
      free(A);
      free(B);
      free(C);
    }

    short iterations() { return 1;}

    int n, dim;
    Interval<T> *A, *B, *C, alpha, beta;
};
BENCH_F_F(SerialBench, Symm, SerialBenchSymm)


//----------------------- Syrk Bench -----------------------------------------//


template <class T>
class SerialBenchSyrk : public SerialBench  {
   public:

    void begin() {
      n = r->length;
      dim = std::sqrt(n);
      alpha = r->alpha;
      beta = r->beta;
      A = (Interval<T>*) malloc(n*sizeof(Interval<T>));
      B = (Interval<T>*) malloc(n*sizeof(Interval<T>));
      acopy(n, r->x, A);
      acopy(n, r->y, B);
    }

    void run() {
      syrk( 'u', dim, dim, alpha, beta, A, B);
    }

    void end() {
      free(A);
      free(B);
    }

    short iterations() { return 1;}

    int n, dim;
    Interval<T> *A, *B, alpha, beta;
};
BENCH_F_F(SerialBench, Syrk, SerialBenchSyrk)


//----------------------- Syr2k Bench -----------------------------------------//


template <class T>
class SerialBenchSyr2k : public SerialBench  {
   public:

    void begin() {
      n = r->length;
      dim = std::sqrt(n);
      alpha = r->alpha;
      beta = r->beta;
      A = (Interval<T>*) malloc(n*sizeof(Interval<T>));
      B = (Interval<T>*) malloc(n*sizeof(Interval<T>));
      C = (Interval<T>*) malloc(n*sizeof(Interval<T>));
      acopy(n, r->x, A);
      acopy(n, r->y, B);
      acopy(n, r->y, C);
    }

    void run() {
      syr2k('u', dim, dim, alpha, beta, A, B, C);
    }

    void end() {
      free(A);
      free(B);
      free(C);
    }

    short iterations() { return 1;}

    int n, dim;
    Interval<T> *A, *B, *C, alpha, beta;
};
BENCH_F_F(SerialBench, Syr2k, SerialBenchSyr2k)


//----------------------- Trmm Bench -----------------------------------------//


template <class T>
class SerialBenchTrmm : public SerialBench  {
   public:

    void begin() {
      n = r->length;
      dim = std::sqrt(n);
      alpha = r->alpha;
      A = (Interval<T>*) malloc(n*sizeof(Interval<T>));
      B = (Interval<T>*) malloc(n*sizeof(Interval<T>));
      acopy(n, r->x, A);
      acopy(n, r->y, B);
    }

    void run() {
      trmm('l', 'u', dim, dim, alpha, A, B);
    }

    void end() {
      free(A);
      free(B);
    }

    short iterations() { return 1;}

    int n, dim;
    Interval<T> *A, *B, alpha;
};
BENCH_F_F(SerialBench, Trmm, SerialBenchTrmm)


//----------------------- Trsm Bench -----------------------------------------//


template <class T>
class SerialBenchTrsm : public SerialBench  {
   public:

    void begin() {
      n = r->length;
      dim = std::sqrt(n);
      alpha = r->alpha;
      A = (Interval<T>*) malloc(n*sizeof(Interval<T>));
      B = (Interval<T>*) malloc(n*sizeof(Interval<T>));
      acopy(n, r->x, A);
      acopy(n, r->y, B);
      for(int i = 0; i < dim; ++i)
        A[i*dim+i] = intlag::abs(A[i*dim+i]) + 0.1;
    }

    void run() {
      trsm('l', 'u', dim, dim, alpha, A, B);
    }

    void end() {

      for(int i = 0; i < dim*dim; ++i)
        EXPECT_FALSE(isnan(B[i].inf()));

      free(A);
      free(B);
    }

    short iterations() { return 1;}

    int n, dim;
    Interval<T> *A, *B, alpha;
};
BENCH_F_F(SerialBench, Trsm, SerialBenchTrsm)



} // namespace bench
} // namespace intlag

#endif



