#include "hip/hip_runtime.h"

//------------------------------------------------------------------------------
// Copyright (c) 2014 Ademar Marques Lacerda Filho
//
// This Source Code Form is subject to the terms of the Mozilla Public
// License, v. 2.0. If a copy of the MPL was not distributed with this
// file, You can obtain one at http://mozilla.org/MPL/2.0/.
//------------------------------------------------------------------------------


#ifndef CUDA_BENCH_H
#define CUDA_BENCH_H

#include <cmath>

#include "aux/case.h"
#include "aux/reference.h"
#include "../include/blas/serial_blas.h"
#include "../include/blas/cuda_blas.h"
#include "gtest/gtest.h"



//FIXME: Separate References by template, think of it as a fixture



namespace intlag {
namespace bench {

//TODO: Compartimentalize this functions
template <class T>
void EXPECT_INTERVAL_EQ(CudaInterval<T> x, CudaInterval<T> y) {};

template<>
void EXPECT_INTERVAL_EQ(CudaInterval<float> x, CudaInterval<float> y) {
  EXPECT_FLOAT_EQ(x.inf(), y.inf());
  EXPECT_FLOAT_EQ(x.sup(), y.sup());
}

template<>
void EXPECT_INTERVAL_EQ(CudaInterval<double> x, CudaInterval<double> y) {
  EXPECT_DOUBLE_EQ(x.inf(), y.inf());
  EXPECT_DOUBLE_EQ(x.sup(), y.sup());
}


//----------------------- Fixture --------------------------------------------//
class CudaBench : public BenchTest {
  public:

    CudaBench() {
      r = Reference::getInstance();
    }
    virtual ~CudaBench() {}

    void SetUp() {
    }
    void TearDown() {}

    Reference* r;
};


//----------------------- Scal Bench -----------------------------------------//


template <class T>
class CudaBenchScal : public CudaBench  {
   public:

    void begin() {
      n = r->length;
      alpha = r->alpha;
      x = (CudaInterval<T>*) malloc(n*sizeof(CudaInterval<T>));
      y = (CudaInterval<T>*) malloc(n*sizeof(CudaInterval<T>));
      acopy(n, r->x, x);
    }

    void run() {
      DeviceData< CudaInterval<T> > dx(n, x);
      CudaGeneral::scal(n, alpha, dx.data());
      dx.toHost(y);
    }

    void end() {
      free(x);
      free(y);
    }

    int n;
    CudaInterval<T> *x, *y, alpha;
};
BENCH_FD_F(CudaBench, Scal, CudaBenchScal)


//----------------------- AXPY Bench -----------------------------------------//


template <class T>
class CudaBenchAXPY : public CudaBench  {
   public:

    void begin() {
      n = r->length;
      alpha = r->alpha;
      x = (CudaInterval<T>*) malloc(n*sizeof(CudaInterval<T>));
      y = (CudaInterval<T>*) malloc(n*sizeof(CudaInterval<T>));
      acopy(n, r->x, x);
      acopy(n, r->y, y);
    }

    void run() {
      DeviceData< CudaInterval<T> > dx(n, x);
      DeviceData< CudaInterval<T> > dy(n, y);
      CudaGeneral::axpy(n, alpha, dx.data(), dy.data());
      dy.toHost(y);
    }

    void end() {
      free(x);
      free(y);
    }

    int n;
    CudaInterval<T> *x, *y, alpha;
};
BENCH_FD_F(CudaBench, AXPY, CudaBenchAXPY)


//----------------------- Gemv Bench -----------------------------------------//


template <class T>
class CudaBenchGemv : public CudaBench  {
   public:

    void begin() {
      n = r->length;
      dim = std::sqrt(n);
      alpha = r->alpha;
      beta = r->beta;
      A = (CudaInterval<T>*) malloc(n*sizeof(CudaInterval<T>));
      x = (CudaInterval<T>*) malloc(dim*sizeof(CudaInterval<T>));
      y = (CudaInterval<T>*) malloc(dim*sizeof(CudaInterval<T>));
      acopy(n, r->x, A);
      acopy(dim, r->y, x);
      acopy(dim, r->y, y);
    }

    void run() {
      DeviceData< CudaInterval<T> > dA(n, A);
      DeviceData< CudaInterval<T> > dx(dim, x);
      DeviceData< CudaInterval<T> > dy(dim, y);
      CudaGeneral::gemv(dim, dim, alpha, beta, dA.data(), dx.data(), dy.data());
      dy.toHost(y);
    }

    void end() {
      free(A);
      free(x);
      free(y);
    }

    int n, dim;
    CudaInterval<T> *A, *x, *y, alpha, beta;
};
BENCH_FD_F(CudaBench, Gemv, CudaBenchGemv)


//----------------------- Gemv Trans Bench -----------------------------------//


template <class T>
class CudaBenchGemvTrans : public CudaBench  {
   public:

    void begin() {
      n = r->length;
      dim = std::sqrt(n);
      alpha = r->alpha;
      beta = r->beta;
      A = (CudaInterval<T>*) malloc(n*sizeof(CudaInterval<T>));
      x = (CudaInterval<T>*) malloc(dim*sizeof(CudaInterval<T>));
      y = (CudaInterval<T>*) malloc(dim*sizeof(CudaInterval<T>));
      acopy(n, r->x, A);
      acopy(dim, r->y, x);
      acopy(dim, r->y, y);
    }

    void run() {
      CudaTrans::gemv(dim, dim, alpha, beta, A, x, y);
    }

    void end() {
      free(A);
      free(x);
      free(y);
    }

    int n, dim;
    CudaInterval<T> *A, *x, *y, alpha, beta;
};
BENCH_FD_F(CudaBench, GemvTrans, CudaBenchGemvTrans)


//----------------------- Gemm Bench -----------------------------------------//


template <class T>
class CudaBenchGemm : public CudaBench  {
   public:

    void begin() {
      n = r->length;
      dim = std::sqrt(n);
      alpha = r->alpha;
      beta = r->beta;
      A = (CudaInterval<T>*) malloc(n*sizeof(CudaInterval<T>));
      B = (CudaInterval<T>*) malloc(n*sizeof(CudaInterval<T>));
      C = (CudaInterval<T>*) malloc(n*sizeof(CudaInterval<T>));
      acopy(n, r->x, A);
      acopy(n, r->y, B);
      acopy(n, r->y, C);
    }

    void run() {
      DeviceData< CudaInterval<T> > dA(n, A);
      DeviceData< CudaInterval<T> > dB(n, B);
      DeviceData< CudaInterval<T> > dC(n, C);
      CudaGeneral::gemm(dim, dim, dim, alpha, beta, dA.data(), dB.data(), dC.data());
      dC.toHost(C);
    }

    void end() {
      free(A);
      free(B);
      free(C);
    }

    short iterations() { return 10;}

    int n, dim;
    CudaInterval<T> *A, *B, *C, alpha, beta;
};
BENCH_FD_F(CudaBench, Gemm, CudaBenchGemm)


//----------------------- Gemm Trans Bench -----------------------------------//


template <class T>
class CudaBenchGemmTrans : public CudaBench  {
   public:

    void begin() {
      n = r->length;
      dim = std::sqrt(n);
      alpha = r->alpha;
      beta = r->beta;
      A = (CudaInterval<T>*) malloc(n*sizeof(CudaInterval<T>));
      B = (CudaInterval<T>*) malloc(n*sizeof(CudaInterval<T>));
      C = (CudaInterval<T>*) malloc(n*sizeof(CudaInterval<T>));
      acopy(n, r->x, A);
      acopy(n, r->y, B);
      acopy(n, r->y, C);
    }

    void run() {
      CudaTrans::gemm(dim, dim, dim, alpha, beta, A, B, C);
    }

    void end() {
      free(A);
      free(B);
      free(C);
    }

    short iterations() { return 10;}

    int n, dim;
    CudaInterval<T> *A, *B, *C, alpha, beta;
};
BENCH_FD_F(CudaBench, GemmTrans, CudaBenchGemmTrans)



} // namespace bench
} // namespace intlag

#endif



