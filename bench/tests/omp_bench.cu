#include "hip/hip_runtime.h"

//------------------------------------------------------------------------------
// Copyright (c) 2014 Ademar Marques Lacerda Filho
//
// This Source Code Form is subject to the terms of the Mozilla Public
// License, v. 2.0. If a copy of the MPL was not distributed with this
// file, You can obtain one at http://mozilla.org/MPL/2.0/.
//------------------------------------------------------------------------------


#ifndef OMP_BENCH_H
#define OMP_BENCH_H


#include <omp.h>

#include "aux/case.h"
#include "aux/reference.h"

#include "../include/blas/serial_blas.h"
#include "../include/blas/omp_blas.h"
#include "gtest/gtest.h"

namespace intlag {
namespace bench {


//----------------------- Fixture --------------------------------------------//

class OMPBench : public BenchTest {
  public:

    OMPBench() {
	    //omp_set_num_threads(16);
      r = Reference::getInstance();
    }
    virtual ~OMPBench() {}

    void SetUp() {}
    void TearDown() {}

    Reference* r;
};


//----------------------- Scal Bench -----------------------------------------//

template <class T>
class OMPBenchScal : public OMPBench  {
   public:

    void begin() {


      n = r->length;
      alpha = r->alpha;
      x = new Interval<T> [n];
      acopy(n, r->x, x);
    }

    void run() {
      omp::scal(n, alpha, x);
    }

    void end() {
      delete[] x;
    }

    int n;
    Interval<T> *x, alpha;
};
BENCH_F_F(OMPBench, Scal, OMPBenchScal)


//----------------------- AXPY Bench -----------------------------------------//

template <class T>
class OMPBenchAXPY : public OMPBench  {
   public:

    void begin() {
      n = r->length;
      alpha = r->alpha;
      x = new Interval<T> [n];
      y = new Interval<T> [n];
      acopy(n, r->x, x);
      acopy(n, r->y, y);
    }

    void run() {
      omp::axpy(n, alpha, x, y);
    }

    void end() {
      delete[] x;
      delete[] y;
    }

    int n;
    Interval<T> *x, *y, alpha;
};
BENCH_F_F(OMPBench, AXPY, OMPBenchAXPY)


//----------------------- ASum Bench -----------------------------------------//

template <class T>
class OMPBenchASum : public OMPBench  {
   public:

    void begin() {
      n = r->length;
      x = new Interval<T> [n];
      acopy(n, r->x, x);
    }

    void run() {
      omp::asum(n, &ret, x);
    }

    void end() {
      delete[] x;
    }

    int n;
    Interval<T> *x, ret;
};
BENCH_F_F(OMPBench, Asum, OMPBenchASum)


//----------------------- Dot Bench -----------------------------------------//

template <class T>
class OMPBenchDot : public OMPBench  {
   public:

    void begin() {
      n = r->length;
      x = new Interval<T> [n];
      y = new Interval<T> [n];
      acopy(n, r->x, x);
      acopy(n, r->y, y);
    }

    void run() {
      omp::dot(n, &ret, x, y);
    }

    void end() {
      delete[] x;
      delete[] y;
    }

    int n;
    Interval<T> *x, *y, ret;
};
BENCH_F_F(OMPBench, Dot, OMPBenchDot)


//----------------------- Norm2 Bench -----------------------------------------//

template <class T>
class OMPBenchNorm2 : public OMPBench  {
   public:

    void begin() {
      n = r->length;
      x = new Interval<T> [n];
      acopy(n, r->x, x);
    }

    void run() {
      omp::norm2(n, &ret, x);
    }

    void end() {
      delete[] x;
    }

    int n;
    Interval<T> *x, ret;
};
BENCH_F_F(OMPBench, Norm2, OMPBenchNorm2)


//----------------------- Rot Bench -----------------------------------------//

template <class T>
class OMPBenchRot : public OMPBench  {
   public:

    void begin() {
      n = r->length;
      x = new Interval<T> [n];
      acopy(n, r->x, x);
      y = new Interval<T> [n];
      acopy(n, r->y, y);
    }

    void run() {
      omp::rot(n, x, y, 0.5, 0.5);
    }

    void end() {
      delete[] x;
      delete[] y;
    }

    int n;
    Interval<T> *x, *y;
};
BENCH_F_F(OMPBench, Rot, OMPBenchRot)


//----------------------- Rotm Bench -----------------------------------------//

template <class T>
class OMPBenchRotm : public OMPBench  {
   public:

    void begin() {
      h[0] = -1; h[1] = 0.5; h[2] = 0.6; h[3] = 0.7; h[4] = 0.8;
      n = r->length;
      x = new Interval<T> [n];
      acopy(n, r->x, x);
      y = new Interval<T> [n];
      acopy(n, r->y, y);
    }

    void run() {
      omp::rotm(n, x, y, h);
    }

    void end() {
      delete[] x;
      delete[] y;
    }

    int n;
    T h[5];
    Interval<T> *x, *y;
};
BENCH_F_F(OMPBench, Rotm, OMPBenchRotm)


//----------------------- Ger Bench -----------------------------------------//


template <class T>
class OMPBenchGer : public OMPBench  {
   public:

    void begin() {
      n = r->length;
      dim = std::sqrt(n);
      alpha = r->alpha;
      A = new Interval<T> [n];
      x = new Interval<T> [dim];
      y = new Interval<T> [dim];
      acopy(n, r->x, A);
      acopy(dim, r->x, x);
      acopy(dim, r->y, y);
    }

    void run() {
      omp::ger(dim, dim, alpha, x, y, A);
    }

    void end() {
      delete[] A;
      delete[] x;
      delete[] y;
    }

    int n, dim;
    Interval<T> *A, *x, *y, alpha, beta;
};
BENCH_F_F(OMPBench, Ger, OMPBenchGer)


//----------------------- Syr Bench -----------------------------------------//


template <class T>
class OMPBenchSyr : public OMPBench  {
   public:

    void begin() {
      n = r->length;
      dim = std::sqrt(n);
      alpha = r->alpha;
      A = new Interval<T> [n];
      x = new Interval<T> [dim];
      acopy(n, r->x, A);
      acopy(dim, r->x, x);
    }

    void run() {
      omp::syr('u', dim, alpha, x, A);
    }

    void end() {
      delete[] A;
      delete[] x;
    }

    int n, dim;
    Interval<T> *A, *x, alpha, beta;
};
BENCH_F_F(OMPBench, Syr, OMPBenchSyr)


//----------------------- Syr2 Bench -----------------------------------------//


template <class T>
class OMPBenchSyr2 : public OMPBench  {
   public:

    void begin() {
      n = r->length;
      dim = std::sqrt(n);
      alpha = r->alpha;
      A = new Interval<T> [n];
      x = new Interval<T> [dim];
      y = new Interval<T> [dim];
      acopy(n, r->x, A);
      acopy(dim, r->x, x);
      acopy(dim, r->y, y);
    }

    void run() {
      omp::syr2('u', dim, alpha, x, y, A);
    }

    void end() {
      delete[] A;
      delete[] x;
      delete[] y;
    }

    int n, dim;
    Interval<T> *A, *x, *y, alpha, beta;
};
BENCH_F_F(OMPBench, Syr2, OMPBenchSyr2)


//----------------------- Spr Bench -----------------------------------------//


template <class T>
class OMPBenchSpr : public OMPBench  {
   public:

    void begin() {
      n = r->length;
      dim = std::sqrt(n);
      alpha = r->alpha;
      A = new Interval<T> [n];
      x = new Interval<T> [dim];
      acopy(n, r->x, A);
      acopy(dim, r->x, x);
    }

    void run() {
      omp::spr('u', dim, alpha, x, A);
    }

    void end() {
      delete[] A;
      delete[] x;
    }

    int n, dim;
    Interval<T> *A, *x, alpha, beta;
};
BENCH_F_F(OMPBench, Spr, OMPBenchSpr)


//----------------------- Spr2 Bench -----------------------------------------//


template <class T>
class OMPBenchSpr2 : public OMPBench  {
   public:

    void begin() {
      n = r->length;
      dim = std::sqrt(n);
      alpha = r->alpha;
      A = new Interval<T> [n];
      x = new Interval<T> [dim];
      y = new Interval<T> [dim];
      acopy(n, r->x, A);
      acopy(dim, r->x, x);
      acopy(dim, r->y, y);
    }

    void run() {
      omp::spr2('u', dim, alpha, x, y, A);
    }

    void end() {
      delete[] A;
      delete[] x;
      delete[] y;
    }

    int n, dim;
    Interval<T> *A, *x, *y, alpha, beta;
};
BENCH_F_F(OMPBench, Spr2, OMPBenchSpr2)


//----------------------- Gemv Bench -----------------------------------------//


template <class T>
class OMPBenchGemv : public OMPBench  {
   public:

    void begin() {
      n = r->length;
      dim = std::sqrt(n);
      alpha = r->alpha;
      beta = r->beta;
      A = new Interval<T> [n];
      x = new Interval<T> [dim];
      y = new Interval<T> [dim];
      acopy(n, r->x, A);
      acopy(dim, r->x, x);
      acopy(dim, r->y, y);
    }

    void run() {
      omp::gemv(dim, dim, alpha, beta, A, x, y);
    }

    void end() {
      delete[] A;
      delete[] x;
      delete[] y;
    }

    int n, dim;
    Interval<T> *A, *x, *y, alpha, beta;
};
BENCH_F_F(OMPBench, Gemv, OMPBenchGemv)


//----------------------- Gbmv Bench -----------------------------------------//


template <class T>
class OMPBenchGbmv : public OMPBench  {
   public:

    void begin() {
      n = r->length;
      dim = std::sqrt(n);
      band = dim/4;
      alpha = r->alpha;
      beta = r->beta;
      A = new Interval<T> [n];
      x = new Interval<T> [dim];
      y = new Interval<T> [dim];
      acopy(n, r->x, A);
      acopy(dim, r->x, x);
      acopy(dim, r->y, y);
    }

    void run() {
      omp::gbmv(dim, dim, band, band, alpha, beta, A, x, y);
    }

    void end() {
      delete[] A;
      delete[] x;
      delete[] y;
    }

    int n, dim, band;
    Interval<T> *A, *x, *y, alpha, beta;
};
BENCH_F_F(OMPBench, Gbmv, OMPBenchGbmv)


//----------------------- Symv Bench -----------------------------------------//


template <class T>
class OMPBenchSymv : public OMPBench  {
   public:

    void begin() {
      n = r->length;
      dim = std::sqrt(n);
      alpha = r->alpha;
      beta = r->beta;
      A = new Interval<T> [n];
      x = new Interval<T> [dim];
      y = new Interval<T> [dim];
      acopy(n, r->x, A);
      acopy(dim, r->x, x);
      acopy(dim, r->y, y);
    }

    void run() {
      omp::symv('u', dim, alpha, beta, A, x, y);
    }

    void end() {
      delete[] A;
      delete[] x;
      delete[] y;
    }

    int n, dim;
    Interval<T> *A, *x, *y, alpha, beta;
};
BENCH_F_F(OMPBench, Symv, OMPBenchSymv)


//----------------------- Sbmv Bench -----------------------------------------//


template <class T>
class OMPBenchSbmv : public OMPBench  {
   public:

    void begin() {
      n = r->length;
      dim = std::sqrt(n);
      band = dim/4;
      alpha = r->alpha;
      beta = r->beta;
      A = new Interval<T> [n];
      x = new Interval<T> [dim];
      y = new Interval<T> [dim];
      acopy(n, r->x, A);
      acopy(dim, r->x, x);
      acopy(dim, r->y, y);
    }

    void run() {
      omp::sbmv('u', dim, band, alpha, beta, A, x, y);
    }

    void end() {
      delete[] A;
      delete[] x;
      delete[] y;
    }

    int n, dim, band;
    Interval<T> *A, *x, *y, alpha, beta;
};
BENCH_F_F(OMPBench, Sbmv, OMPBenchSbmv)


//----------------------- Spmv Bench -----------------------------------------//


template <class T>
class OMPBenchSpmv : public OMPBench  {
   public:

    void begin() {
      n = r->length;
      dim = std::sqrt(n);
      alpha = r->alpha;
      beta = r->beta;
      A = new Interval<T> [n];
      x = new Interval<T> [dim];
      y = new Interval<T> [dim];
      acopy(n, r->x, A);
      acopy(dim, r->x, x);
      acopy(dim, r->y, y);
    }

    void run() {
      omp::spmv('u', dim, alpha, beta, A, x, y);
    }

    void end() {
      delete[] A;
      delete[] x;
      delete[] y;
    }

    int n, dim;
    Interval<T> *A, *x, *y, alpha, beta;
};
BENCH_F_F(OMPBench, Spmv, OMPBenchSpmv)


//----------------------- Trmv Bench -----------------------------------------//


template <class T>
class OMPBenchTrmv : public OMPBench  {
   public:

    void begin() {
      n = r->length;
      dim = std::sqrt(n);
      A = new Interval<T> [n];
      x = new Interval<T> [dim];
      acopy(n, r->y, A);
      acopy(dim, r->x, x);
    }

    void run() {
      omp::trmv('u', dim, A, x);
    }

    void end() {
      delete[] A;
      delete[] x;
    }

    int n, dim;
    Interval<T> *A, *x;
};
BENCH_F_F(OMPBench, Trmv, OMPBenchTrmv)


//----------------------- Tbmv Bench -----------------------------------------//


template <class T>
class OMPBenchTbmv : public OMPBench  {
   public:

    void begin() {
      n = r->length;
      dim = std::sqrt(n);
      band = dim/4;
      A = new Interval<T> [n];
      x = new Interval<T> [dim];
      acopy(n, r->y, A);
      acopy(dim, r->x, x);
    }

    void run() {
      omp::tbmv('u', dim, band, A, x);
    }

    void end() {
      delete[] A;
      delete[] x;
    }

    int n, dim, band;
    Interval<T> *A, *x;
};
BENCH_F_F(OMPBench, Tbmv, OMPBenchTbmv)


//----------------------- Tpmv Bench -----------------------------------------//


template <class T>
class OMPBenchTpmv : public OMPBench  {
   public:

    void begin() {
      n = r->length;
      dim = std::sqrt(n);
      A = new Interval<T> [n];
      x = new Interval<T> [dim];
      acopy(n, r->y, A);
      acopy(dim, r->x, x);
    }

    void run() {
      omp::tpmv('u', dim, A, x);
    }

    void end() {
      delete[] A;
      delete[] x;
    }

    int n, dim;
    Interval<T> *A, *x;
};
BENCH_F_F(OMPBench, Tpmv, OMPBenchTpmv)


//----------------------- Trsv Bench -----------------------------------------//


template <class T>
class OMPBenchTrsv : public OMPBench  {
   public:

    void begin() {
      n = r->length;
      dim = std::sqrt(n);
      A = new Interval<T> [n];
      x = new Interval<T> [dim];
      acopy(n, r->y, A);
      acopy(dim, r->x, x);
      for(int i = 0; i < dim; ++i)
        A[i*dim+i] = intlag::abs(A[i*dim+i]) + 0.1;
    }

    void run() {
      omp::trsv('u', dim, A, x);
    }

    void end() {

      for(int i = 0; i < dim; ++i)
        EXPECT_FALSE(isnan(x[i].inf()));

      delete[] A;
      delete[] x;
    }

    int n, dim;
    Interval<T> *A, *x;
};
BENCH_F_F(OMPBench, Trsv, OMPBenchTrsv)


//----------------------- Tbsv Bench -----------------------------------------//


template <class T>
class OMPBenchTbsv : public OMPBench  {
   public:

    void begin() {
      n = r->length;
      dim = std::sqrt(n);
      band = dim/4;
      A = new Interval<T> [n];
      x = new Interval<T> [dim];
      acopy(n, r->y, A);
      acopy(dim, r->x, x);
      for(int i = 0; i < dim; ++i)
        A[i*band+i] = intlag::abs(A[i*band+i]) + 0.1;
    }

    void run() {
      omp::tbsv('u', dim, band, A, x);
    }

    void end() {

      for(int i = 0; i < dim; ++i)
        EXPECT_FALSE(isnan(x[i].inf()));

      delete[] A;
      delete[] x;
    }

    int n, dim, band;
    Interval<T> *A, *x;
};
BENCH_F_F(OMPBench, Tbsv, OMPBenchTbsv)


//----------------------- Tpsv Bench -----------------------------------------//


template <class T>
class OMPBenchTpsv : public OMPBench  {
   public:

    void begin() {
      n = r->length;
      dim = std::sqrt(n);
      A = new Interval<T> [n];
      x = new Interval<T> [dim];
      acopy(n, r->y, A);
      acopy(dim, r->x, x);
      for(int i = 0; i < dim; ++i)
        A[INDEX_TRIAN_UP(dim, i, i)] = intlag::abs(A[INDEX_TRIAN_UP(dim, i, i)]) + 0.1;
    }

    void run() {
      omp::tpsv('u', dim, A, x);
    }

    void end() {

      for(int i = 0; i < dim; ++i)
        EXPECT_FALSE(isnan(x[i].inf()));

      delete[] A;
      delete[] x;
    }

    int n, dim;
    Interval<T> *A, *x;
};
BENCH_F_F(OMPBench, Tpsv, OMPBenchTpsv)


//----------------------- Gemm Bench -----------------------------------------//


template <class T>
class OMPBenchGemm : public OMPBench  {
   public:

    void begin() {
      n = r->length;
      dim = std::sqrt(n);
      alpha = r->alpha;
      beta = r->beta;
      A = (Interval<T>*) malloc(n*sizeof(Interval<T>));
      B = (Interval<T>*) malloc(n*sizeof(Interval<T>));
      C = (Interval<T>*) malloc(n*sizeof(Interval<T>));
      acopy(n, r->x, A);
      acopy(n, r->y, B);
      acopy(n, r->y, C);
    }

    void run() {
      omp::gemm(dim, dim, dim, alpha, beta, A, B, C);
    }

    void end() {
      free(A);
      free(B);
      free(C);
    }

    short iterations() { return 1;}

    int n, dim;
    Interval<T> *A, *B, *C, alpha, beta;
};
BENCH_F_F(OMPBench, Gemm, OMPBenchGemm)


//----------------------- Symm Bench -----------------------------------------//


template <class T>
class OMPBenchSymm : public OMPBench  {
   public:

    void begin() {
      n = r->length;
      dim = std::sqrt(n);
      alpha = r->alpha;
      beta = r->beta;
      A = (Interval<T>*) malloc(n*sizeof(Interval<T>));
      B = (Interval<T>*) malloc(n*sizeof(Interval<T>));
      C = (Interval<T>*) malloc(n*sizeof(Interval<T>));
      acopy(n, r->x, A);
      acopy(n, r->y, B);
      acopy(n, r->y, C);
    }

    void run() {
      omp::symm('l', 'u', dim, dim, alpha, beta, A, B, C);
    }

    void end() {
      free(A);
      free(B);
      free(C);
    }

    short iterations() { return 1;}

    int n, dim;
    Interval<T> *A, *B, *C, alpha, beta;
};
BENCH_F_F(OMPBench, Symm, OMPBenchSymm)


//----------------------- Syrk Bench -----------------------------------------//


template <class T>
class OMPBenchSyrk : public OMPBench  {
   public:

    void begin() {
      n = r->length;
      dim = std::sqrt(n);
      alpha = r->alpha;
      beta = r->beta;
      A = (Interval<T>*) malloc(n*sizeof(Interval<T>));
      B = (Interval<T>*) malloc(n*sizeof(Interval<T>));
      acopy(n, r->x, A);
      acopy(n, r->y, B);
    }

    void run() {
      omp::syrk( 'u', dim, dim, alpha, beta, A, B);
    }

    void end() {
      free(A);
      free(B);
    }

    short iterations() { return 1;}

    int n, dim;
    Interval<T> *A, *B, alpha, beta;
};
BENCH_F_F(OMPBench, Syrk, OMPBenchSyrk)


//----------------------- Syr2k Bench -----------------------------------------//


template <class T>
class OMPBenchSyr2k : public OMPBench  {
   public:

    void begin() {
      n = r->length;
      dim = std::sqrt(n);
      alpha = r->alpha;
      beta = r->beta;
      A = (Interval<T>*) malloc(n*sizeof(Interval<T>));
      B = (Interval<T>*) malloc(n*sizeof(Interval<T>));
      C = (Interval<T>*) malloc(n*sizeof(Interval<T>));
      acopy(n, r->x, A);
      acopy(n, r->y, B);
      acopy(n, r->y, C);
    }

    void run() {
      omp::syr2k('u', dim, dim, alpha, beta, A, B, C);
    }

    void end() {
      free(A);
      free(B);
      free(C);
    }

    short iterations() { return 1;}

    int n, dim;
    Interval<T> *A, *B, *C, alpha, beta;
};
BENCH_F_F(OMPBench, Syr2k, OMPBenchSyr2k)


//----------------------- Trmm Bench -----------------------------------------//


template <class T>
class OMPBenchTrmm : public OMPBench  {
   public:

    void begin() {
      n = r->length;
      dim = std::sqrt(n);
      alpha = r->alpha;
      A = (Interval<T>*) malloc(n*sizeof(Interval<T>));
      B = (Interval<T>*) malloc(n*sizeof(Interval<T>));
      acopy(n, r->x, A);
      acopy(n, r->y, B);
    }

    void run() {
      omp::trmm('l', 'u', dim, dim, alpha, A, B);
    }

    void end() {
      free(A);
      free(B);
    }

    short iterations() { return 1;}

    int n, dim;
    Interval<T> *A, *B, alpha;
};
BENCH_F_F(OMPBench, Trmm, OMPBenchTrmm)


//----------------------- Trsm Bench -----------------------------------------//


template <class T>
class OMPBenchTrsm : public OMPBench  {
   public:

    void begin() {
      n = r->length;
      dim = std::sqrt(n);
      alpha = r->alpha;
      A = (Interval<T>*) malloc(n*sizeof(Interval<T>));
      B = (Interval<T>*) malloc(n*sizeof(Interval<T>));
      acopy(n, r->x, A);
      acopy(n, r->y, B);
      for(int i = 0; i < dim; ++i)
        A[i*dim+i] = intlag::abs(A[i*dim+i]) + 0.1;
    }

    void run() {
      omp::trsm('l', 'u', dim, dim, alpha, A, B);
    }

    void end() {

      for(int i = 0; i < dim*dim; ++i)
        EXPECT_FALSE(isnan(B[i].inf()));

      free(A);
      free(B);
    }

    short iterations() { return 1;}

    int n, dim;
    Interval<T> *A, *B, alpha;
};
BENCH_F_F(OMPBench, Trsm, OMPBenchTrsm)



} // namespace bench
} // namespace intlag

#endif



