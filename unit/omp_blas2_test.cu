#include "hip/hip_runtime.h"

//------------------------------------------------------------------------------
// Copyright (c) 2014 Ademar Marques Lacerda Filho
//
// This Source Code Form is subject to the terms of the Mozilla Public
// License, v. 2.0. If a copy of the MPL was not distributed with this
// file, You can obtain one at http://mozilla.org/MPL/2.0/.
//------------------------------------------------------------------------------

#ifndef OMP_BLAS2_TEST_CU
#define OMP_BLAS2_TEST_CU

#include <math.h>
#include "blas/serial_blas.h"
#include "blas/omp_blas.h"
#include "aux/test_interval.h"
#include "aux/reference.h"
#include "aux/test_interval.h"
#include "gtest/gtest.h"


#include "blas/cuda_blas.h"

#include "aux/ulp.h"


using namespace intlag;


class OMPBlasTest : public ::testing::Test {
  protected:

    OMPBlasTest() {
      ref = Reference::getInstance();
    }
    virtual ~OMPBlasTest(){}

    virtual void SetUp()
    {
      for(int i = 0; i < 9; ++i) {
          a[i] = Interval<float>(i);
          b[i] = Interval<float>(-i, i);
          c[i] = Interval<float>(i-5, i+i*i);
      }
      for(int i = 0; i < 10000; ++i) {
          x[i] = Interval<float>(i);
          y[i] = Interval<float>(-i, i);
          z[i] = Interval<float>(i-5000, i+i*i);
      }

      for(int i = 0; i < 9; ++i) {
          da[i] = Interval<double>(i);
          db[i] = Interval<double>(-i, i);
          dc[i] = Interval<double>(i-5, i+i*i);
      }
      for(int i = 0; i < 10000; ++i) {
          dx[i] = Interval<double>(i);
          dy[i] = Interval<double>(-i, i);
          dz[i] = Interval<double>(i-5000, i+i*i);
      }
    }
    virtual void TearDown() {}

    Reference* ref;

    Interval<float> a[9], b[9], c[9];
    Interval<float> x[10000], y[10000], z[10000];
    Interval<double> da[9], db[9], dc[9], dx[10000], dy[10000], dz[10000];
};


/******************************** Tests ***************************************/


TEST_F(OMPBlasTest, Trans) {

  Interval<double> x[10000], xtrans[10000];
  omp::acopy(10000, ref->x, x);
  omp::acopy(10000, ref->x, xtrans);

  omp::trans(34, 21, xtrans);
  for(int i = 0; i < 34; ++i)    
		for (int j = 0; j < 21; ++j)
      EXPECT_FINTERVAL_EQ(x[i*21+j], xtrans[j*34+i]);
}


TEST_F(OMPBlasTest, DiagonalUnit) {

  Interval<double> x[10000];
  omp::acopy(10000, ref->x, x);

  omp::diagonal_unit(33, 34, x);
  for(int i = 0; i < 33; ++i)
    EXPECT_FINTERVAL_EQ(x[i*34+i], 1.0);
}


TEST_F(OMPBlasTest, Ger) {

  Interval<float> out[] = {1.0, 2.0, 3.0, 4.0, 5.0, 6.0};

  omp::ger(2, 3, -2.2, a, b, out);
  EXPECT_FINTERVAL_EQ(out[0], Interval<float>(1.0));
  EXPECT_FINTERVAL_EQ(out[1], Interval<float>(2.0));
  EXPECT_FINTERVAL_EQ(out[2], Interval<float>(3.0));
  EXPECT_FINTERVAL_EQ(out[3], Interval<float>(4.0));
  EXPECT_FINTERVAL_EQ(out[4], Interval<float>(2.8, 7.2));
  EXPECT_FINTERVAL_EQ(out[5], Interval<float>(1.6, 10.4));
}


TEST_F(OMPBlasTest, Syr) {

  Interval<float> out_up[] = {1.0, 2.0, 3.0, 2.0, 5.0, 6.0, 5.0, 6.0, 9.0};
  Interval<float> out_sym_up[] = {1.0, 2.0, 3.0, -400.0, 5.0, 6.0, -600.0, -300.0, 9.0};
  Interval<float> out_down[] = {1.0, 4.0, 7.0, 4.0, 5.0, 8.0, 7.0, 8.0, 9.0};
  Interval<float> out_sym_down[] = {1.0, -4.0, 27.0, 4.0, 5.0, -18.0, 7.0, 8.0, 9.0};

  ger(3, 3, 2.2, a, a, out_up);
  omp::syr('U', 3, 2.2, a, out_sym_up);
  for (int i = 0; i < 3; ++i)
    for (int j = i; j < 3; ++j)
      EXPECT_FINTERVAL_EQ(out_up[i*3+j], out_sym_up[i*3+j]);

  ger(3, 3, 2.2, a, a, out_down);
  omp::syr('l', 3, 2.2, a, out_sym_down);
  for (int i = 0; i < 3; ++i)
    for (int j = 0; j <= i; ++j)
      EXPECT_FINTERVAL_EQ(out_down[i*3+j], out_sym_down[i*3+j]);
}


TEST_F(OMPBlasTest, Syr2) {

  Interval<float> zeroes[] = {0.0, 0.0, 0.0, 0.0, 0.0, 0.0, 0.0, 0.0, 0.0};
  Interval<float> zeroes2[] = {0.0, 0.0, 0.0, 0.0, 0.0, 0.0, 0.0, 0.0, 0.0};
  Interval<float> out_up[] = {1.0, 2.0, 3.0, 2.0, 5.0, 6.0, 5.0, 6.0, 9.0};
  Interval<float> out_sym_up[] = {1.0, 2.0, 3.0, -400.0, 5.0, 6.0, -600.0, -300.0, 9.0};
  Interval<float> out_down[] = {1.0, 4.0, 7.0, 4.0, 5.0, 8.0, 7.0, 8.0, 9.0};
  Interval<float> out_sym_down[] = {1.0, -4.0, 27.0, 4.0, 5.0, -18.0, 7.0, 8.0, 9.0};

  ger(3, 3, 2.2, a, b, zeroes);
  ger(3, 3, 2.2, b, a, out_up);
  axpy(9, 1.0, zeroes, out_up);
  omp::syr2('U', 3, 2.2, a, b, out_sym_up);
  for (int i = 0; i < 3; ++i)
    for (int j = i; j < 3; ++j)
      EXPECT_FINTERVAL_EQ(out_up[i*3+j], out_sym_up[i*3+j]);

  ger(3, 3, 2.2, a, b, zeroes2);
  ger(3, 3, 2.2, a, b, out_down);
  axpy(9, 1.0, zeroes2, out_down);
  omp::syr2('l', 3, 2.2, a, b, out_sym_down);
  for (int i = 0; i < 3; ++i)
    for (int j = 0; j <= i; ++j)
      EXPECT_FINTERVAL_EQ(out_down[i*3+j], out_sym_down[i*3+j]);
}


TEST_F(OMPBlasTest, Spr) {

  Interval<float> out_up[] = {1.0, 2.0, 3.0, 2.0, 5.0, 6.0, 5.0, 6.0, 9.0};
  Interval<float> out_sym_up[] = {1.0, 2.0, 3.0, 5.0, 6.0, 9.0};
  Interval<float> out_down[] = {1.0, 4.0, 7.0, 4.0, 5.0, 8.0, 7.0, 8.0, 9.0};
  Interval<float> out_sym_down[] = {1.0, 4.0, 5.0, 7.0, 8.0, 9.0};

  ger(3, 3, 2.2, a, a, out_up);
  omp::spr('u', 3, 2.2, a, out_sym_up);
  EXPECT_FINTERVAL_EQ(out_up[0], out_sym_up[0]);
  EXPECT_FINTERVAL_EQ(out_up[1], out_sym_up[1]);
  EXPECT_FINTERVAL_EQ(out_up[2], out_sym_up[2]);
  EXPECT_FINTERVAL_EQ(out_up[4], out_sym_up[3]);
  EXPECT_FINTERVAL_EQ(out_up[5], out_sym_up[4]);
  EXPECT_FINTERVAL_EQ(out_up[8], out_sym_up[5]);

  ger(3, 3, 2.2, a, a, out_down);
  omp::spr('L', 3, 2.2, a, out_sym_down);
  EXPECT_FINTERVAL_EQ(out_down[0], out_sym_down[0]);
  EXPECT_FINTERVAL_EQ(out_down[3], out_sym_down[1]);
  EXPECT_FINTERVAL_EQ(out_down[4], out_sym_down[2]);
  EXPECT_FINTERVAL_EQ(out_down[6], out_sym_down[3]);
  EXPECT_FINTERVAL_EQ(out_down[7], out_sym_down[4]);
  EXPECT_FINTERVAL_EQ(out_down[8], out_sym_up[5]);
}


TEST_F(OMPBlasTest, Spr2) {

  Interval<float> zeroes[] = {0.0, 0.0, 0.0, 0.0, 0.0, 0.0, 0.0, 0.0, 0.0};
  Interval<float> zeroes2[] = {0.0, 0.0, 0.0, 0.0, 0.0, 0.0, 0.0, 0.0, 0.0};
  Interval<float> out_up[] = {1.0, 2.0, 3.0, 2.0, 5.0, 6.0, 5.0, 6.0, 9.0};
  Interval<float> out_sym_up[] = {1.0, 2.0, 3.0, 5.0, 6.0, 9.0};
  Interval<float> out_down[] = {1.0, 4.0, 7.0, 4.0, 5.0, 8.0, 7.0, 8.0, 9.0};
  Interval<float> out_sym_down[] = {1.0, 4.0, 5.0, 7.0, 8.0, 9.0};

  ger(3, 3, 2.2, a, b, zeroes);
  ger(3, 3, 2.2, b, a, out_up);
  axpy(9, 1.0, zeroes, out_up);
  omp::spr2('U', 3, 2.2, a, b, out_sym_up);
  EXPECT_FINTERVAL_EQ(out_up[0], out_sym_up[0]);
  EXPECT_FINTERVAL_EQ(out_up[1], out_sym_up[1]);
  EXPECT_FINTERVAL_EQ(out_up[2], out_sym_up[2]);
  EXPECT_FINTERVAL_EQ(out_up[4], out_sym_up[3]);
  EXPECT_FINTERVAL_EQ(out_up[5], out_sym_up[4]);
  EXPECT_FINTERVAL_EQ(out_up[8], out_sym_up[5]);

  ger(3, 3, 2.2, a, b, zeroes2);
  ger(3, 3, 2.2, a, b, out_down);
  axpy(9, 1.0, zeroes2, out_down);
  omp::spr2('l', 3, 2.2, a, b, out_sym_down);
  EXPECT_FINTERVAL_EQ(out_down[0], out_sym_down[0]);
  EXPECT_FINTERVAL_EQ(out_down[3], out_sym_down[1]);
  EXPECT_FINTERVAL_EQ(out_down[4], out_sym_down[2]);
  EXPECT_FINTERVAL_EQ(out_down[6], out_sym_down[3]);
  EXPECT_FINTERVAL_EQ(out_down[7], out_sym_down[4]);
  EXPECT_FINTERVAL_EQ(out_down[8], out_sym_up[5]);

}


TEST_F(OMPBlasTest, Gemv) {

  Interval<float> out[3];

  omp::acopy(3, b, out);
  omp::gemv(1, 1, -2.5, 3.5, c, a, out);
  EXPECT_FINTERVAL_EQ(out[0], -2.5*c[0]*a[0]+3.5*b[0]);

  omp::acopy(3, b, out);
  omp::gemv(2, 2, Interval<float>(-2,2), 0.0, c, a, out);
  EXPECT_FINTERVAL_EQ(out[0], Interval<float>(-8.0,8.0));
  EXPECT_FINTERVAL_EQ(out[1], Interval<float>(-24.0,24.0));

  omp::gemv(3, 3, 2.0, 3.0, c, a, b);
  EXPECT_FINTERVAL_EQ(b[0], Interval<float>(-20.0, 28.0));
  EXPECT_FINTERVAL_EQ(b[1], Interval<float>(-5.0, 163.0));
  EXPECT_FINTERVAL_EQ(b[2], Interval<float>(10.0, 406.0));
  
  Interval<double> x[10000], y[10000], z[10000], w[10000];
  omp::acopy(10000, ref->x, x);
  omp::acopy(10000, ref->y, y);
  omp::acopy(10000, ref->x, z);
  omp::acopy(10000, ref->x, w);

  gemv(340, 27, 4.444, -3.5, x, y, w);
  omp::gemv(340, 27, 4.444, -3.5, x, y, z);

  for(int i = 0; i < 340; ++i)
    EXPECT_FINTERVAL_EQ(z[i], w[i]);
  for(int i = 340; i < 10000; ++i)
    EXPECT_FINTERVAL_EQ(z[i], x[i]);
};


TEST_F(OMPBlasTest, Gbmv) {

  Interval<double> x[10000], A[10000], y1[10000], y2[10000];
  omp::acopy(10000, ref->x, x);
  omp::acopy(10000, ref->y, y1);
  omp::acopy(10000, ref->y, y2);
  omp::acopy(10000, ref->x, A);

  gbmv(33, 22, 5, 8, 22.5, 4.2, A, x, y1);
  omp::gbmv(33, 22, 5, 8, 22.5, 4.2, A, x, y2);

  for(int i = 0; i < 33; ++i)
    EXPECT_DINTERVAL_EQ(y1[i], y2[i]);
}


TEST_F(OMPBlasTest, Symv) {

  Interval<double> x[10000], A[10000], y1[10000], y2[10000];
  omp::acopy(10000, ref->x, x);
  omp::acopy(10000, ref->y, y1);
  omp::acopy(10000, ref->y, y2);
  omp::acopy(10000, ref->x, A);

  symv('U', 23, 2.5, 33.54, A, x, y1);
  omp::symv('u', 23, 2.5, 33.54, A, x, y2);
  for(int i = 0; i < 23; ++i)
    EXPECT_DINTERVAL_EQ(y1[i], y2[i]);

  symv('l', 23, 2.5, 33.54, A, x, y1);
  omp::symv('L', 23, 2.5, 33.54, A, x, y2);
  for(int i = 0; i < 23; ++i)
    EXPECT_DINTERVAL_EQ(y1[i], y2[i]);

}


TEST_F(OMPBlasTest, Sbmv) {

  Interval<double> x[10000], A[10000], y1[10000], y2[10000];
  omp::acopy(10000, ref->x, x);
  omp::acopy(10000, ref->y, y1);
  omp::acopy(10000, ref->y, y2);
  omp::acopy(10000, ref->x, A);

  sbmv('U', 22, 5, 2.5, 33.54, A, x, y1);
  omp::sbmv('u', 22, 5, 2.5, 33.54, A, x, y2);
  for(int i = 0; i < 22; ++i)
    EXPECT_DINTERVAL_EQ(y1[i], y2[i]);

  sbmv('L', 22, 5, 2.5, 33.54, A, x, y1);
  omp::sbmv('l', 22, 5, 2.5, 33.54, A, x, y2);
  for(int i = 0; i < 22; ++i)
    EXPECT_DINTERVAL_EQ(y1[i], y2[i]);
}


TEST_F(OMPBlasTest, Spmv) {
  Interval<double> x[10000], A[10000], y1[10000], y2[10000];
  omp::acopy(10000, ref->x, x);
  omp::acopy(10000, ref->y, y1);
  omp::acopy(10000, ref->y, y2);
  omp::acopy(10000, ref->x, A);

  spmv('U', 22, 2.5, 33.54, A, x, y1);
  omp::spmv('u', 22, 2.5, 33.54, A, x, y2);
  for(int i = 0; i < 22; ++i)
    EXPECT_DINTERVAL_EQ(y1[i], y2[i]);

  spmv('L', 22, 2.5, 33.54, A, x, y1);
  omp::spmv('l', 22, 2.5, 33.54, A, x, y2);
  for(int i = 0; i < 22; ++i)
    EXPECT_DINTERVAL_EQ(y1[i], y2[i]);
}


TEST_F(OMPBlasTest, Trmv) {
  Interval<double> A[10000], x1[10000], x2[10000];
  omp::acopy(10000, ref->x, x1);
  omp::acopy(10000, ref->x, x2);
  omp::acopy(10000, ref->y, A);

  trmv('U', 22, A, x1);
  omp::trmv('u', 22, A, x2);
  for(int i = 0; i < 22; ++i)
    EXPECT_DINTERVAL_EQ(x1[i], x2[i]);

  trmv('L', 23, A, x1);
  omp::trmv('l', 23, A, x2);
  for(int i = 0; i < 23; ++i)
    EXPECT_DINTERVAL_EQ(x1[i], x2[i]);
}


TEST_F(OMPBlasTest, Tbmv) {
  Interval<double> A[10000], x1[10000], x2[10000];
  omp::acopy(10000, ref->x, x1);
  omp::acopy(10000, ref->x, x2);
  omp::acopy(10000, ref->y, A);

  tbmv('U', 22, 3, A, x1);
  omp::tbmv('u', 22, 3, A, x2);
  for(int i = 0; i < 22; ++i)
    EXPECT_DINTERVAL_EQ(x1[i], x2[i]);

  tbmv('L', 23, 5, A, x1);
  omp::tbmv('l', 23, 5, A, x2);
  for(int i = 0; i < 23; ++i)
    EXPECT_DINTERVAL_EQ(x1[i], x2[i]);
}


TEST_F(OMPBlasTest, Tpmv) {
  Interval<double> A[10000], x1[10000], x2[10000];
  omp::acopy(10000, ref->x, x1);
  omp::acopy(10000, ref->x, x2);
  omp::acopy(10000, ref->y, A);

  tpmv('U', 22, A, x1);
  omp::tpmv('u', 22, A, x2);
  for(int i = 0; i < 22; ++i)
    EXPECT_DINTERVAL_EQ(x1[i], x2[i]);

  tpmv('L', 23, A, x1);
  omp::tpmv('l', 23, A, x2);
  for(int i = 0; i < 23; ++i)
    EXPECT_DINTERVAL_EQ(x1[i], x2[i]);
}


TEST_F(OMPBlasTest, Trsv) {
  Interval<double> A[10000], x1[10000], x2[10000];
  omp::acopy(10000, ref->x, x1);
  omp::acopy(10000, ref->x, x2);
  omp::acopy(10000, ref->y, A);

  // A diagonal must not contain 0
  for(int i = 0; i < 23; ++i)
    A[23*i+i] = Interval<double>(i+3.0, i+30.0);

  trsv('U', 23, A, x1);
  omp::trsv('u', 23, A, x2);
  for(int i = 0; i < 23; ++i)
    EXPECT_DINTERVAL_EQ(x1[i], x2[i]);


  for(int i = 0; i < 24; ++i)
    A[24*i+i] = Interval<double>(-(i+34.0), -(i+4.0));
  trsv('L', 24, A, x1);
  omp::trsv('l', 24, A, x2);
  for(int i = 0; i < 24; ++i)
    EXPECT_DINTERVAL_EQ(x1[i], x2[i]);
}


TEST_F(OMPBlasTest, Tbsv) {
  Interval<double> A[10000], x1[10000], x2[10000];
  omp::acopy(10000, ref->x, x1);
  omp::acopy(10000, ref->x, x2);
  omp::acopy(10000, ref->y, A);

  // A diagonal must not contain 0
  for(int i = 0; i < 23; ++i)
    A[5*i+0] = Interval<double>(i+3.0, i+30.0);

  tbsv('U', 23, 4, A, x1);
  omp::tbsv('u', 23, 4, A, x2);
  for(int i = 0; i < 23; ++i)
    EXPECT_DINTERVAL_EQ(x1[i], x2[i]);


  for(int i = 0; i < 24; ++i)
    A[5*i+4] = Interval<double>(-(i+34.0), -(i+4.0));
  tbsv('L', 24, 4, A, x1);
  omp::tbsv('l', 24, 4, A, x2);
  for(int i = 0; i < 24; ++i)
    EXPECT_DINTERVAL_EQ(x1[i], x2[i]);
}


TEST_F(OMPBlasTest, Tpsv) {

  Interval<double> A[10000], x1[10000], x2[10000];
  omp::acopy(10000, ref->x, x1);
  omp::acopy(10000, ref->x, x2);
  omp::acopy(10000, ref->y, A);

  // A diagonal must not contain 0
  for(int i = 0; i < 23; ++i)
    A[INDEX_TRIAN_UP(23, i, i)] = Interval<double>(-(i+34.0), -(i+4.0));

  tpsv('U', 23, A, x1);
  omp::tpsv('u', 23, A, x2);
  for(int i = 0; i < 23; ++i)
    EXPECT_DINTERVAL_EQ(x1[i], x2[i]);


  for(int i = 0; i < 24; ++i)
    A[INDEX_TRIAN_DOWN(24, i, i)] = Interval<double>(i+3.0, i+30.0);
  tpsv('L', 24, A, x1);
  omp::tpsv('l', 24, A, x2);
  for(int i = 0; i < 24; ++i)
    EXPECT_DINTERVAL_EQ(x1[i], x2[i]);

}


#endif



