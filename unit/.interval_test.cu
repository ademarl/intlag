#include "hip/hip_runtime.h"

//------------------------------------------------------------------------------
// Copyright (c) 2013 Ademar Marques Lacerda Filho
//
// This Source Code Form is subject to the terms of the Mozilla Public
// License, v. 2.0. If a copy of the MPL was not distributed with this
// file, You can obtain one at http://mozilla.org/MPL/2.0/.
//------------------------------------------------------------------------------

#ifndef INTERVAL_TEST
#define INTERVAL_TEST

#include "stdio.h"
#include "gtest/gtest.h"

#include "gpu_interval.h"
#include "gpu_interval/cuda_interval_lib.h"

using namespace intlag;

typedef interval_gpu<float> If_t;
typedef interval_gpu<double> Id_t;



//////////////////////////////////////////////////////////// Kernels ///////////
template <class T>
__global__ void add(T a, T b, T *c){
	*c = a + b;
}

template <class T>
__global__ void scalL(double alpha, T x, T *y){
	*y = alpha*x;
}

template <class T>
__global__ void scalR(double alpha, T x, T *y){
	*y =x*alpha;
}

template <class T>
__global__ void fma(double alpha, T x, T y, T* z){
	*z = fma(alpha, x, y);
}

template <class T>
__global__ void sqrt_test(interval_gpu<T> x, interval_gpu<T>* y){
		// check for signal of upper and lower extremes
		*y = sqrt(x);
}
///////////////////////////////////////////////////////////// ~Kernels /////////


////////////////////////////////////////////////////////// CUDA Functions //////
template <class T>
void sum_intervals(T a, T b, T *c){

	T *d_c;
	CHECKED_CALL( hipMalloc((void**)&d_c, sizeof(T)) );
	add<<<1,1>>>(a, b, d_c);
	CHECKED_CALL( hipMemcpy(c, d_c, sizeof(T), hipMemcpyDeviceToHost) );
	CHECKED_CALL( hipFree(d_c) );
}

template <class T>
void scalL_intervals(double alpha, T x, T *y){

	T *d_y;
	CHECKED_CALL( hipMalloc((void**)&d_y, sizeof(T)) );
	scalL<<<1,1>>>(alpha, x, d_y);
	CHECKED_CALL( hipMemcpy(y, d_y, sizeof(T), hipMemcpyDeviceToHost) );
	CHECKED_CALL( hipFree(d_y) );
}


template <class T>
void scalR_intervals(double alpha, T x, T *y){

	T *d_y;
	CHECKED_CALL( hipMalloc((void**)&d_y, sizeof(T)) );
	scalR<<<1,1>>>(alpha, x, d_y);
	CHECKED_CALL( hipMemcpy(y, d_y, sizeof(T), hipMemcpyDeviceToHost) );
	CHECKED_CALL( hipFree(d_y) );
}

template <class T>
void fma_intervals(double alpha, T a, T b, T *c){

	T *d_c;
	CHECKED_CALL( hipMalloc((void**)&d_c, sizeof(T)) );
	fma<<<1,1>>>(alpha, a, b, d_c);
	CHECKED_CALL( hipMemcpy(c, d_c, sizeof(T), hipMemcpyDeviceToHost) );
	CHECKED_CALL( hipFree(d_c) );
}

template <class T>
void sqrt_intervals(T a, T *b){

	T *d_b;
	CHECKED_CALL( hipMalloc((void**)&d_b, sizeof(T)) );
	sqrt_test<<<1,1>>>(a, d_b);
	CHECKED_CALL( hipMemcpy(b, d_b, sizeof(T), hipMemcpyDeviceToHost) );
	CHECKED_CALL( hipFree(d_b) );
}

////////////////////////////////////////////////////////// ~CUDA Functions /////

class IntervalTest : public ::testing::Test{

	protected:

	virtual void SetUp(){
		x = Id_t(-2.1, 3.5);		
		y = Id_t(3.0, 5.5);
		z = Id_t();
	}

	Id_t x, y, z;
};

TEST_F(IntervalTest, Create) {

	EXPECT_DOUBLE_EQ(x.lower(), -2.1);
	EXPECT_DOUBLE_EQ(x.upper(), 3.5);

};


TEST_F(IntervalTest, Addition) {

	sum_intervals(x, y, &z);
	EXPECT_DOUBLE_EQ(0.9, z.lower());
	EXPECT_DOUBLE_EQ(9.0, z.upper());

};

TEST_F(IntervalTest, ScalMultL) {

	scalL_intervals(0.5, x, &z);
	EXPECT_DOUBLE_EQ(-1.05, z.lower());
	EXPECT_DOUBLE_EQ(1.75, z.upper());

};

TEST_F(IntervalTest, ScalMultR) {

	scalR_intervals(-0.5, x, &z);
	EXPECT_DOUBLE_EQ(-1.75, z.lower());
	EXPECT_DOUBLE_EQ(1.05, z.upper());

};

TEST_F(IntervalTest, FMA) {

	fma_intervals(-0.5, x, y, &z);
	EXPECT_DOUBLE_EQ(1.25, z.lower());
	EXPECT_DOUBLE_EQ(6.55, z.upper());

};

TEST_F(IntervalTest, SQRT) {

	sqrt_intervals(y, &z);
	EXPECT_NEAR(1.732, z.lower(), 0.1);
	EXPECT_NEAR(2.345, z.upper(), 0.1);

};

#endif
