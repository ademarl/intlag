#include "hip/hip_runtime.h"

//------------------------------------------------------------------------------
// Copyright (c) 2014 Ademar Marques Lacerda Filho
//
// This Source Code Form is subject to the terms of the Mozilla Public
// License, v. 2.0. If a copy of the MPL was not distributed with this
// file, You can obtain one at http://mozilla.org/MPL/2.0/.
//------------------------------------------------------------------------------

#ifndef ROUNDER_TEST_CU
#define ROUNDER_TEST_CU


#include <math.h>
#include "interval/rounder.h"
#include "gtest/gtest.h"

#include "aux/ulp.h"


using namespace intlag;

#define FLT_PRECISION 5e-6
#define DBL_PRECISION 5e-15


class RounderTest : public ::testing::Test {
  protected:

    RounderTest() {
      old_status = getRoundMode();
    }
    virtual ~RounderTest()
    {
    }

    virtual void SetUp()
    {
      x = 2.222222222;
      y = 3.333333333;
      a = 2.22222222222222222222;
      b = 3.33333333333333333333;

      fesetround(FE_DOWNWARD);
      sum = x+y;
      sub = x-y;
      mul = x*y;
      div = x/y;
      ma = fma(2.0f, x, y);
      sq   = sqrt(x);
      dsum = a+b;
      dsub = a-b;
      dmul = a*b;
      ddiv = a/b;
      dma = fma(2.0, a, b);
      dsq   = sqrt(a);
      setRoundMode(old_status);
    }
    virtual void TearDown()
    {
      setRoundMode(old_status);
    }


    short old_status;
    float x, y, sum, sub, mul, div, ma, sq;
    double a, b, dsum, dsub, dmul, ddiv, dma, dsq;
};




/********************************Tests ************/
TEST_F(RounderTest, Fenv) {

  setRoundMode(RoundStatus::up);
  ASSERT_NEAR(x+y, nextlarger(sum), FLT_PRECISION);
  ASSERT_NEAR(a+b, nextlarger(dsum), DBL_PRECISION);

  setRoundMode(RoundStatus::down);
  ASSERT_NEAR(x+y, sum, DBL_PRECISION);
  ASSERT_NEAR(a+b, dsum, DBL_PRECISION);
};

TEST_F(RounderTest, NaN) {

  ASSERT_TRUE(isnan(Rounder<float>::nan()));
  ASSERT_TRUE(isnan(Rounder<float>::nan()));
};

TEST_F(RounderTest, Sum) {

  ASSERT_NEAR(Rounder<float>::add_up(x, y), nextlarger(sum), FLT_PRECISION);
  ASSERT_NEAR(Rounder<float>::add_down(x, y), sum, FLT_PRECISION);
  ASSERT_NEAR(Rounder<double>::add_up(a,b), nextlarger(dsum), DBL_PRECISION);
  ASSERT_NEAR(Rounder<double>::add_down(a,b), dsum, DBL_PRECISION);
};

TEST_F(RounderTest, Sub) {

  ASSERT_NEAR(Rounder<float>::sub_up(x, y), nextlarger(sub), FLT_PRECISION);
  ASSERT_NEAR(Rounder<float>::sub_down(x, y), sub, FLT_PRECISION);
  ASSERT_NEAR(Rounder<double>::sub_up(a,b), nextlarger(dsub), DBL_PRECISION);
  ASSERT_NEAR(Rounder<double>::sub_down(a,b), dsub, DBL_PRECISION);
};

TEST_F(RounderTest, Mul) {

  ASSERT_NEAR(Rounder<float>::mul_up(x, y), nextlarger(mul), FLT_PRECISION);
  ASSERT_NEAR(Rounder<float>::mul_down(x, y), mul, FLT_PRECISION);
  ASSERT_NEAR(Rounder<double>::mul_up(a,b), nextlarger(dmul), DBL_PRECISION);
  ASSERT_NEAR(Rounder<double>::mul_down(a,b), dmul, DBL_PRECISION);
};

TEST_F(RounderTest, Div) {

  ASSERT_NEAR(Rounder<float>::div_up(x, y), nextlarger(div), FLT_PRECISION);
  ASSERT_NEAR(Rounder<float>::div_down(x, y), div, FLT_PRECISION);
  ASSERT_NEAR(Rounder<double>::div_up(a,b), nextlarger(ddiv), DBL_PRECISION);
  ASSERT_NEAR(Rounder<double>::div_down(a,b), ddiv, DBL_PRECISION);
};


TEST_F(RounderTest, Fma) {

  ASSERT_NEAR(Rounder<float>::fma_up(2.0, x, y), nextlarger(ma), FLT_PRECISION);
  ASSERT_NEAR(Rounder<float>::fma_down(2.0, x, y), ma, FLT_PRECISION);
  ASSERT_NEAR(Rounder<double>::fma_up(2.0, a,b), nextlarger(dma), DBL_PRECISION);
  ASSERT_NEAR(Rounder<double>::fma_down(2.0, a,b), dma, DBL_PRECISION);
};

TEST_F(RounderTest, Sqrt) {

  ASSERT_NEAR(Rounder<float>::sqrt_up(x), nextlarger(sq), FLT_PRECISION);
  ASSERT_NEAR(Rounder<float>::sqrt_down(x), sq, FLT_PRECISION);
  ASSERT_NEAR(Rounder<double>::sqrt_up(a), nextlarger(dsq), DBL_PRECISION);
  ASSERT_NEAR(Rounder<double>::sqrt_down(a), dsq, DBL_PRECISION);
};


#endif
