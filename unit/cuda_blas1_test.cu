#include "hip/hip_runtime.h"

//------------------------------------------------------------------------------
// Copyright (c) 2014 Ademar Marques Lacerda Filho
//
// This Source Code Form is subject to the terms of the Mozilla Public
// License, v. 2.0. If a copy of the MPL was not distributed with this
// file, You can obtain one at http://mozilla.org/MPL/2.0/.
//------------------------------------------------------------------------------


#ifndef CUDA_BLAS1_TEST
#define CUDA_BLAS1_TEST

#include "gtest/gtest.h"
#include "aux/reference.h"
#include "aux/test_interval.h"

#include "blas/serial_blas.h"
#include "blas/omp_blas.h"
#include "blas/cuda_blas.h"

using namespace intlag;


class CudaBlasTest : public ::testing::Test{

	protected:

    CudaBlasTest () {
      ref = Reference::getInstance();
    }

	  virtual void SetUp() {

		  x[0] = CudaInterval<double>(1.1, 2.2);
		  x[1] = CudaInterval<double>(3.3, 4.4);
		  x[2] = CudaInterval<double>(5.5, 6.6);

		  y[0] = CudaInterval<double>(10.1, 20.1);
		  y[1] = CudaInterval<double>(30.2, 40.2);
		  y[2] = CudaInterval<double>(-500.0, -300.0);

	    A[0] = CudaInterval<double>(1.0);
      A[1] = CudaInterval<double>(1.0);
      A[2] = CudaInterval<double>(2.0);
      A[3] = CudaInterval<double>(4.0);
	    B[0] = CudaInterval<double>(1.0, 2.0);
      B[1] = CudaInterval<double>(3.0, 4.0);
	    C[0] = CudaInterval<double>(1.0, 2.0);
      C[1] = CudaInterval<double>(3.0, 4.0);
	  }

	  virtual void TearDown() { }

	  CudaInterval<double> x[3], y[3], z[3], A[4], B[3], C[3], r;
    Reference* ref;
};


TEST_F(CudaBlasTest, ACopy) {

	acopy(3, x, z);

	EXPECT_DOUBLE_EQ(1.1, z[0].inf());
	EXPECT_DOUBLE_EQ(3.3, z[1].inf());
	EXPECT_DOUBLE_EQ(5.5, z[2].inf());
	EXPECT_DOUBLE_EQ(2.2, z[0].sup());
	EXPECT_DOUBLE_EQ(4.4, z[1].sup());
	EXPECT_DOUBLE_EQ(6.6, z[2].sup());
}


TEST_F(CudaBlasTest, Swap) {

	swap(3, x, y);

	EXPECT_DOUBLE_EQ(1.1, y[0].inf());
	EXPECT_DOUBLE_EQ(3.3, y[1].inf());
	EXPECT_DOUBLE_EQ(5.5, y[2].inf());
	EXPECT_DOUBLE_EQ(2.2, y[0].sup());
	EXPECT_DOUBLE_EQ(4.4, y[1].sup());
	EXPECT_DOUBLE_EQ(6.6, y[2].sup());

	EXPECT_DOUBLE_EQ(10.1, x[0].inf());
	EXPECT_DOUBLE_EQ(30.2, x[1].inf());
	EXPECT_DOUBLE_EQ(-500.0, x[2].inf());
	EXPECT_DOUBLE_EQ(20.1, x[0].sup());
	EXPECT_DOUBLE_EQ(40.2, x[1].sup());
	EXPECT_DOUBLE_EQ(-300.0, x[2].sup());
}


TEST_F(CudaBlasTest, Scal) {

	CudaGeneralManaged::scal(3, (double)2, x);

	EXPECT_DOUBLE_EQ(2*1.1, x[0].inf());
	EXPECT_DOUBLE_EQ(2*3.3, x[1].inf());
	EXPECT_DOUBLE_EQ(2*5.5, x[2].inf());
	EXPECT_DOUBLE_EQ(2*2.2, x[0].sup());
	EXPECT_DOUBLE_EQ(2*4.4, x[1].sup());
	EXPECT_DOUBLE_EQ(2*6.6, x[2].sup());
}


TEST_F(CudaBlasTest, AXPY) {

	CudaGeneralManaged::axpy(3, 2, x, y);

	EXPECT_DOUBLE_EQ(2*1.1+10.1, y[0].inf());
	EXPECT_DOUBLE_EQ(2*3.3+30.2, y[1].inf());
	EXPECT_DOUBLE_EQ(2*5.5-500, y[2].inf());
	EXPECT_DOUBLE_EQ(2*2.2+20.1, y[0].sup());
	EXPECT_DOUBLE_EQ(2*4.4+40.2, y[1].sup());
	EXPECT_DOUBLE_EQ(2*6.6-300, y[2].sup());
}


TEST_F(CudaBlasTest, ASum) {

	CudaGeneralManaged::asum(3, &r, x);

	EXPECT_DOUBLE_EQ(9.9, r.inf());
	EXPECT_DOUBLE_EQ(13.2, r.sup());
}


TEST_F(CudaBlasTest, Dot) {

	CudaGeneralManaged::dot(3, &r, x, y);

	//EXPECT_DOUBLE_EQ(0, r.inf());
	EXPECT_DOUBLE_EQ(2.2*20.1 + 4.4*40.2 + 500*6.6, r.sup());
}


TEST_F(CudaBlasTest, Norm2) {

	Interval<double> x1[99], r1;
	CudaInterval<double> x2[99];
	acopy(99, ref->x, x1);
	acopy(99, ref->x, x2);

	norm2(99, &r1, x1);
	CudaGeneralManaged::norm2(99, &r, x2);

	// SQRT is not exact
	EXPECT_NEAR(r1.inf(), r.inf(), 1e-6);
	EXPECT_NEAR(r1.sup(), r.sup(), 1e-6);
}


TEST_F(CudaBlasTest, Rot) {

  Interval<double> x1[34], y1[34];
  CudaInterval<double> x2[34], y2[34];
  acopy(34, ref->x, x1);
  acopy(34, ref->y, y1);
  acopy(34, ref->x, x2);
  acopy(34, ref->y, y2);

  omp::rot(34, x1, y1, 0.5, 0.6);
  CudaGeneralManaged::rot(34, x2, y2, 0.5, 0.6);
  for(int i = 0; i < 34; ++i)
    EXPECT_DINTERVAL_EQ(x1[i], x2[i]);
}


TEST_F(CudaBlasTest, Rotm) {

  Interval<double> x1[34], y1[34];
  CudaInterval<double> x2[34], y2[34];
  acopy(34, ref->x, x1);
  acopy(34, ref->y, y1);
  acopy(34, ref->x, x2);
  acopy(34, ref->y, y2);

  double H2[] = {-2,15,16,17,-18};
  omp::rotm(34, x1, y1, H2);
  CudaGeneralManaged::rotm(34, x2, y2, H2);
  for(int i = 0; i < 34; ++i)
    EXPECT_DINTERVAL_EQ(x1[i], x2[i]);

  double H1[] = {1,0,0,0,0};
  omp::rotm(34, x1, y1, H1);
  CudaGeneralManaged::rotm(34, x2, y2, H1);
  for(int i = 0; i < 34; ++i)
    EXPECT_DINTERVAL_EQ(x1[i], x2[i]);

  double H0[] = {0,0,0,0,0};
  omp::rotm(34, x1, y1, H0);
  CudaGeneralManaged::rotm(34, x2, y2, H0);
  for(int i = 0; i < 34; ++i)
    EXPECT_DINTERVAL_EQ(x1[i], x2[i]);

  double H[] = {-1,0,0,0,0};
  omp::rotm(34, x1, y1, H);
  CudaGeneralManaged::rotm(34, x2, y2, H);
  for(int i = 0; i < 34; ++i)
    EXPECT_DINTERVAL_EQ(x1[i], x2[i]);
}


#endif



