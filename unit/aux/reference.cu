#include "hip/hip_runtime.h"

//------------------------------------------------------------------------------
// Copyright (c) 2014 Ademar Marques Lacerda Filho
//
// This Source Code Form is subject to the terms of the Mozilla Public
// License, v. 2.0. If a copy of the MPL was not distributed with this
// file, You can obtain one at http://mozilla.org/MPL/2.0/.
//------------------------------------------------------------------------------



#include "reference.h"

// Defines a singleton class with random data inputs and reference results for all test operations


namespace intlag
{


Reference* reference = NULL;

void Reference::setValues(int argc, char** argv, int n){
  // Not thread safe
  if (reference == NULL) {
    if (n == 0 && argc == 1) reference = new Reference();
    else reference = new Reference(argc, argv, n);
  }
}


Reference* Reference::getInstance() {
  return reference;
}


//Generates general inputs
void Reference::general_init(){

      neg = Interval<double>(-3.1, -1.2);
      mid = Interval<double>(-5.1, 5.1);
      pos = Interval<double>(2.2, 4.4);

      zero = Interval<double>(0.0);
      point_pos = Interval<double>(2.0);
      point_neg = Interval<double>(-2.0);

      empty_i = Interval<double>();
      empty_down = Interval<double>(nan(""), 2048.8);
      empty_up = Interval<double>(-3000.8, nan(""));

      for(int i = 0; i < 9; ++i) {
          a[i] = Interval<double>(i);
          b[i] = Interval<double>(-i, i);
          c[i] = Interval<double>(i-5, i+i*i);
      }
}


// Generates a random number in [-MULTIPLIER, MULTIPLIER]
Interval<double> Reference::random_interval() {
  double x, y;
  x = (2*MULTIPLIER*rand())/RAND_MAX - MULTIPLIER;
  y = (2*MULTIPLIER*rand())/RAND_MAX - MULTIPLIER;
  if(x > y){
    double z = x;
    x = y;
    y = z;
  }
  return Interval<double>(x, y);
}


void Reference::random_init(int n){

  length = n;

  alpha = random_interval();
  beta = random_interval();

  x = (Interval<double>*) malloc(length*sizeof(Interval<double>));
  y = (Interval<double>*) malloc(length*sizeof(Interval<double>));
  if (x == NULL || y == NULL) {
    printf("Error, not enough memory for benchmark reference values!\n");
    exit(1);
  }

  for (int i = 0; i < length; ++i){
    x[i]  = random_interval();
    y[i]  = random_interval();
  }
}


void Reference::input_init(std::istream& is) {
  double aux, aux2;

  is >> length;

	is >> aux; is >> aux2;
	alpha = Interval<double>(aux, aux2);
	is >> aux; is >> aux2;
	beta = Interval<double>(aux, aux2);

  x = (Interval<double>*) malloc(length*sizeof(Interval<double>));
  y = (Interval<double>*) malloc(length*sizeof(Interval<double>));
  if (x == NULL || y == NULL) {
    printf("Error, not enough memory for benchmark reference values!\n");
    exit(1);
  }

  for (int i = 0; i < length; ++i){
    is >> aux; is >> aux2;
    x[i] = Interval<double>(aux, aux2);
	}
	for (int i = 0; i < length; ++i){
		is >> aux; is >> aux2;
    y[i] = Interval<double>(aux, aux2);
  }

}


// Initializes GPU by invoking CudaGeneral::scal
void Reference::benchCudaInit(Reference *r) {

  CudaInterval<double> *x = (CudaInterval<double>*) malloc((r->length)*sizeof(CudaInterval<double>));
  CudaInterval<double> alpha = r->alpha;
  acopy(r->length, r->x, x);
  DeviceData< CudaInterval<double> > dx(r->length, x);
  CudaGeneral::scal(r->length, alpha, dx.data());
  dx.toHost(x);
}


} // namespace intlag




