
//------------------------------------------------------------------------------
// Copyright (c) 2014 Ademar Marques Lacerda Filho
//
// This Source Code Form is subject to the terms of the Mozilla Public
// License, v. 2.0. If a copy of the MPL was not distributed with this
// file, You can obtain one at http://mozilla.org/MPL/2.0/.
//------------------------------------------------------------------------------

#ifndef DEVICE_DATA_TEST_CU
#define DEVICE_DATA_TEST_CU


#include "aux/device_data.h"
#include "gtest/gtest.h"

using namespace intlag;

TEST(DeviceDataTest, ToHost) {

	int z[4];
	int y[4] = {2, 3, 4, 110};
	DeviceData<int> x(4, y);
	x.toHost(z);

	EXPECT_EQ(2, z[0]);
	EXPECT_EQ(3, z[1]);
	EXPECT_EQ(4, z[2]);
	EXPECT_EQ(110, z[3]);
};

TEST(DeviceDataTest, Data) {

	double a[4] = {5, 6, 7, 42.42};
	double y[4] = {2.0, 3.0, 4.5, 111.1};
	DeviceData<double> x(4, y);
	CHECKED_CALL( hipMemcpy(a, x.data(), 4*sizeof(double), hipMemcpyDeviceToHost) );

	EXPECT_DOUBLE_EQ(2.0, a[0]);
	EXPECT_DOUBLE_EQ(3.0, a[1]);
	EXPECT_DOUBLE_EQ(4.5, a[2]);
	EXPECT_DOUBLE_EQ(111.1, a[3]);
};

TEST(DeviceDataTest, Swap) {

	double z[20000];
	double a[20000];
  for (int i = 0; i < 20000; ++i)
    a[i] = 2.31*i;

	DeviceData<double> x(20000, a), y(20000);
	swapByReference(x, y);
	y.toHost(z);

	EXPECT_DOUBLE_EQ(0, z[0]);
	EXPECT_DOUBLE_EQ(2.31, z[1]);
	EXPECT_DOUBLE_EQ(4.62, z[2]);
	EXPECT_DOUBLE_EQ(23100, z[10000]);
	EXPECT_DOUBLE_EQ(23102.31, z[10001]);
	EXPECT_DOUBLE_EQ(2.31*19999, z[19999]);
};

#endif
