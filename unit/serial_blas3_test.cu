#include "hip/hip_runtime.h"

//------------------------------------------------------------------------------
// Copyright (c) 2014 Ademar Marques Lacerda Filho
//
// This Source Code Form is subject to the terms of the Mozilla Public
// License, v. 2.0. If a copy of the MPL was not distributed with this
// file, You can obtain one at http://mozilla.org/MPL/2.0/.
//------------------------------------------------------------------------------

#ifndef SERIAL_BLAS3_TEST_CU
#define SERIAL_BLAS3_TEST_CU

#include <math.h>
#include "blas/serial_blas.h"
#include "aux/test_interval.h"
#include "aux/reference.h"
#include "gtest/gtest.h"

#include "aux/ulp.h"


using namespace intlag;


class SerialBlasTest : public ::testing::Test {
  protected:

    SerialBlasTest() {
      r = Reference::getInstance();
    }
    virtual ~SerialBlasTest(){}

    virtual void SetUp()
    {
      for(int i = 0; i < 9; ++i) {
          a[i] = Interval<float>(i);
          b[i] = Interval<float>(-i, i);
          c[i] = Interval<float>(i-5, i+i*i);
      }
      for(int i = 0; i < 10000; ++i) {
          x[i] = Interval<float>(i);
          y[i] = Interval<float>(-i, i);
          z[i] = Interval<float>(i-5000, i+i*i);
      }

      for(int i = 0; i < 9; ++i) {
          da[i] = Interval<double>(i);
          db[i] = Interval<double>(-i, i);
          dc[i] = Interval<double>(i-5, i+i*i);
      }
      for(int i = 0; i < 10000; ++i) {
          dx[i] = Interval<double>(i);
          dy[i] = Interval<double>(-i, i);
          dz[i] = Interval<double>(i-5000, i+i*i);
      }
    }
    virtual void TearDown() {}

    Reference* r;

    Interval<float> a[9], b[9], c[9];
    Interval<float> x[10000], y[10000], z[10000];
    Interval<double> da[9], db[9], dc[9], dx[10000], dy[10000], dz[10000];
};




/******************************** Tests ***************************************/


TEST_F(SerialBlasTest, Gemm) {

  Interval<float> out[4];

  acopy(4, b, out);
  gemm(1, 1, 1, -2.5, 3.5, c, a, out);
  EXPECT_FINTERVAL_EQ(out[0], -2.5*c[0]*a[0]+3.5*b[0]);

  acopy(4, b, out);
  gemm(2, 2, 1, -1.0, -5.5, c, b, out);
  EXPECT_FINTERVAL_EQ(out[0], Interval<float>(0.0));
  EXPECT_FINTERVAL_EQ(out[1], Interval<float>(-10.5,10.5));
  EXPECT_FINTERVAL_EQ(out[2], Interval<float>(-11.0,11.0));
  EXPECT_FINTERVAL_EQ(out[3], Interval<float>(-20.5,20.5));

  gemm(3, 1, 3, 2.0, 3.0, c, a, b);
  EXPECT_FINTERVAL_EQ(b[0], Interval<float>(-20.0, 28.0));
  EXPECT_FINTERVAL_EQ(b[1], Interval<float>(-5.0, 163.0));
  EXPECT_FINTERVAL_EQ(b[2], Interval<float>(10.0, 406.0));

  Interval<double> x[10000], y[10000], z[10000], w[10000];
  acopy(10000, r->x, x);
  acopy(10000, r->y, y);
  acopy(10000, r->x, z);
  acopy(34*21, r->x, w);

  gemm(34, 21, 89, 4.0, -3.5, x, y, z);

	for (int i = 0; i < 34; ++i) 
		for (int j = 0; j < 21; ++j) {
			Interval<double> sum = 0.0;
			for (int k = 0; k < 89; ++k)
				sum += x[i*89+k]*y[k*21+j];
			w[i*21+j] = -3.5*w[i*21+j] + 4.0*sum;
		}

  for(int i = 0; i < 34*21; ++i)
    EXPECT_DINTERVAL_EQ(z[i], w[i]);
};


TEST_F(SerialBlasTest, Symm) {

  Interval<double> x[10000], y[10000], z[10000], w[10000];
  acopy(10000, r->x, x);
  acopy(10000, r->y, y);
  acopy(10000, r->x, z);
  acopy(10000, r->x, w);

  // Left Upper 
  for(int i = 0; i < 34; ++i)    
		for (int j = 0; j < i; ++j)
      x[i*34+j] = x[j*34+i];
      
  gemm(34, 21, 34, 4.0, -3.5, x, y, z);
  symm('l', 'u', 34, 21, 4.0, -3.5, x, y, w);

  for(int i = 0; i < 34*21; ++i) {
    EXPECT_NEAR(z[i].inf(), w[i].inf(), 1e-6);
    EXPECT_NEAR(z[i].sup(), w[i].sup(), 1e-6);
  }

  // Left Lower
  for(int i = 0; i < 33; ++i)    
	  for (int j = 0; j < i; ++j)
      x[i*33+j] = x[j*33+i];
      
  intlag::gemm(33, 21, 33, 4.0, -3.5, x, y, z);
  symm('L', 'l', 33, 21, 4.0, -3.5, x, y, w);

  for(int i = 0; i < 33*21; ++i) {
    EXPECT_NEAR(z[i].inf(), w[i].inf(), 1e-6);
    EXPECT_NEAR(z[i].sup(), w[i].sup(), 1e-6);
  }

  // Right Upper 
  for(int i = 0; i < 33; ++i)    
		for (int j = 0; j < i; ++j)
      x[i*33+j] = x[j*33+i];
      
  intlag::gemm(20, 33, 33, 4.0, -3.5, y, x, z);
  symm('r', 'U', 20, 33, 4.0, -3.5, x, y, w);

  for(int i = 0; i < 33*20; ++i) {
    EXPECT_NEAR(z[i].inf(), w[i].inf(), 1e-6);
    EXPECT_NEAR(z[i].sup(), w[i].sup(), 1e-6);
  }

  // Right Lower 
  for(int i = 0; i < 33; ++i)    
		for (int j = 0; j < i; ++j)
      x[i*33+j] = x[j*33+i];
      
  intlag::gemm(20, 33, 33, 4.0, -3.5, y, x, z);
  symm('R', 'L', 20, 33, 4.0, -3.5, x, y, w);

  for(int i = 0; i < 33*20; ++i) {
    EXPECT_NEAR(z[i].inf(), w[i].inf(), 1e-6);
    EXPECT_NEAR(z[i].sup(), w[i].sup(), 1e-6);
  }
}


TEST_F(SerialBlasTest, Syrk) {

  Interval<float> x[10000], xtrans[10000], z[10000], w[10000];
  acopy(10000, r->x, x);
  acopy(10000, r->x, xtrans);
  acopy(10000, r->y, z);
  acopy(10000, r->y, w);
  trans(2, 1, xtrans);

  // Upper 
  for(int i = 0; i < 2; ++i)    
		for (int j = 0; j < i; ++j) {
      z[i*2+j] = z[j*2+i];
      w[i*2+j] = w[j*2+i];
    }

  syrk('u', 2, 1, 4.0, -3.5, x, w);
  EXPECT_FINTERVAL_EQ(w[0], 4.0*x[0]*x[0]-3.5*z[0]);
  EXPECT_FINTERVAL_EQ(w[1], 4.0*x[0]*x[1]-3.5*z[1]);
  EXPECT_FINTERVAL_EQ(w[3], 4.0*x[1]*x[1]-3.5*z[3]);

  // Lower
  acopy(10000, r->y, z);
  acopy(10000, r->y, w);
  for(int i = 0; i < 2; ++i)    
		for (int j = 0; j < i; ++j) {
      z[i*2+j] = z[j*2+i];
      w[i*2+j] = w[j*2+i];
    }

  syrk('l', 2, 1, 4.0, -3.5, x, w);
  EXPECT_FINTERVAL_EQ(w[0], 4.0*x[0]*x[0]-3.5*z[0]);
  EXPECT_FINTERVAL_EQ(w[2], 4.0*x[0]*x[1]-3.5*z[2]);
  EXPECT_FINTERVAL_EQ(w[3], 4.0*x[1]*x[1]-3.5*z[3]);
}


TEST_F(SerialBlasTest, Syr2k) {

  Interval<float> x[10000], y[10000], z[10000], w[10000];
  acopy(10000, r->x, x);
  acopy(10000, r->y, y);
  acopy(10000, r->y, z);
  acopy(10000, r->y, w);

  // Upper 
  for(int i = 0; i < 2; ++i)    
		for (int j = 0; j < i; ++j) {
      z[i*2+j] = z[j*2+i];
      w[i*2+j] = w[j*2+i];
    }

  syr2k('u', 2, 1, 4.0, -3.5, x, y, w);
  EXPECT_FINTERVAL_EQ(w[0], 4.0*(x[0]*y[0]+y[0]*x[0])-3.5*z[0]);
  EXPECT_FINTERVAL_EQ(w[1], 4.0*(x[0]*y[1]+y[0]*x[1])-3.5*z[1]);
  EXPECT_FINTERVAL_EQ(w[3], 4.0*(x[1]*y[1]+y[1]*x[1])-3.5*z[3]);

  // Lower
  acopy(10000, r->y, z);
  acopy(10000, r->y, w);
  for(int i = 0; i < 2; ++i)    
		for (int j = 0; j < i; ++j) {
      z[i*2+j] = z[j*2+i];
      w[i*2+j] = w[j*2+i];
    }
      
  syr2k('l', 2, 1, 4.0, -3.5, x, y, w);
  EXPECT_FINTERVAL_EQ(w[0], 4.0*(x[0]*y[0]+y[0]*x[0])-3.5*z[0]);
  EXPECT_FINTERVAL_EQ(w[2], 4.0*(x[0]*y[1]+y[0]*x[1])-3.5*z[2]);
  EXPECT_FINTERVAL_EQ(w[3], 4.0*(x[1]*y[1]+y[1]*x[1])-3.5*z[3]);

}


TEST_F(SerialBlasTest, Trmm) {

  Interval<float> x1[] = {1,2,0,3};
  Interval<float> y1[] = {11,22};
  Interval<float> zeroes1[] = {0,0};
  intlag::gemm(2, 1, 2, 4.0, 0.0, x1, y1, zeroes1);
  trmm('l', 'u', 2, 1, 4.0, x1, y1);
  EXPECT_FINTERVAL_EQ(y1[0], zeroes1[0]);
  EXPECT_FINTERVAL_EQ(y1[1], zeroes1[1]);


  Interval<float> x2[10000], y2[10000], z2[10000], zeroes2[10000];
  acopy(10000, r->x, x2);
  acopy(10000, r->y, y2);
  acopy(10000, r->y, z2);
  acopy(10000, r->y, zeroes2);

  // Left Upper 
  for(int i = 0; i < 34; ++i)    
		for (int j = 0; j < i; ++j)
      x2[i*34+j] = 0.0;
      
  gemm(34, 21, 34, 4.0, 0.0, x2, y2, zeroes2);
  trmm('l', 'u', 34, 21, 4.0, x2, z2);

  for (int i = 0; i < 34; ++i)
    for (int j = 0; j < 21; ++j)
      EXPECT_FINTERVAL_EQ(z2[i*21+j], zeroes2[i*21+j]);


  // Left Lower
  acopy(10000, r->x, x2);
  acopy(10000, r->y, y2);
  acopy(10000, r->y, z2);
  for(int i = 0; i < 33; ++i)    
	  for (int j = i+1; j < 33; ++j)
      x2[i*33+j] = 0.0;

  gemm(33, 21, 33, 4.0, 0.0, x2, y2, zeroes2);
  trmm('L', 'l', 33, 21, 4.0, x2, z2);

  for(int i = 0; i < 33; ++i)    
		for (int j = 0; j < 21; ++j)
      EXPECT_FINTERVAL_EQ(z2[i*21+j], zeroes2[i*21+j]);

  // Right Upper 
  acopy(10000, r->x, x2);
  acopy(10000, r->y, y2);
  acopy(10000, r->y, z2);
  for(int i = 0; i < 20; ++i)    
	  for (int j = 0; j < i; ++j)
      x2[i*20+j] = 0.0;

  gemm(33, 20, 20, 4.0, 0.0, y2, x2, zeroes2);
  trmm('R', 'u', 33, 20, 4.0, x2, z2);

  for(int i = 0; i < 33; ++i)    
		for (int j = 0; j < 20; ++j)
      EXPECT_FINTERVAL_EQ(z2[i*20+j], zeroes2[i*20+j]);

  // Right Lower 
  acopy(10000, r->x, x2);
  acopy(10000, r->y, y2);
  acopy(10000, r->y, z2);
  for(int i = 0; i < 20; ++i)    
	  for (int j = i+1; j < 20; ++j)
      x2[i*20+j] = 0.0;

  gemm(33, 20, 20, 4.0, 0.0, y2, x2, zeroes2);
  trmm('R', 'L', 33, 20, 4.0, x2, z2);

  for(int i = 0; i < 33; ++i)    
		for (int j = 0; j < 20; ++j)
      EXPECT_FINTERVAL_EQ(z2[i*20+j], zeroes2[i*20+j]);
}


TEST_F(SerialBlasTest, Trsm) {

  Interval<double> x1[10000], y1[10000], z1[10000];
  acopy(10000, r->x, x1);
  acopy(10000, r->y, y1);
  acopy(10000, r->y, z1);


  // Left Upper

  // positive diagonal needed
  for(int i = 0; i < 32; ++i)
      x1[i*32+i] = Interval<double>(i+1.0, i+20.0);

  //operation
  scal(32, Interval<double>(-2.7, 4.0), y1);
  trsv('U', 32, x1, y1);
  trsm('L', 'U', 32, 1, Interval<double>(-2.7, 4.0), x1, z1);

  for(int i = 0; i < 32; ++i) {
    EXPECT_NEAR(z1[i].inf(), y1[i].inf(), 1e-6);
    EXPECT_NEAR(z1[i].sup(), y1[i].sup(), 1e-6);
  }

  // Left Lower

  // positive diagonal needed
  for(int i = 0; i < 33; ++i)
      x1[i*33+i] = Interval<double>(i+0.1, i+40.0);

  //operation
  scal(33, Interval<double>(2.7, 4.0), y1);
  trsv('l', 33, x1, y1);
  trsm('L', 'l', 33, 1, Interval<double>(2.7, 4.0), x1, z1);

  for(int i = 0; i < 33; ++i) {
    EXPECT_NEAR(z1[i].inf(), y1[i].inf(), 1e-6);
    EXPECT_NEAR(z1[i].sup(), y1[i].sup(), 1e-6);
  }

  // Right Upper

  //operation
  acopy(10000, r->x, x1);
  acopy(10000, r->y, y1);
  acopy(10000, r->y, z1);
  for(int i = 0; i < 2; ++i)
    x1[i*2+i] = Interval<double>(i+0.1, i+4.0);
  trsm('r', 'U', 1, 2, -5.0, x1, z1);
  EXPECT_DINTERVAL_EQ(z1[0], -5.0*y1[0]/x1[0]);
  EXPECT_DINTERVAL_EQ(z1[1], (-5.0*y1[1]-z1[0]*x1[1])/x1[3]);

  // Right Lower

  //operation
  acopy(10000, r->x, x1);
  acopy(10000, r->y, y1);
  acopy(10000, r->y, z1);
  for(int i = 0; i < 2; ++i)
    x1[i*2+i] = Interval<double>(i+0.1, i+4.0);
  trsm('r', 'L', 1, 2, -5.0, x1, z1);
  EXPECT_DINTERVAL_EQ(z1[1], -5.0*y1[1]/x1[3]);
  EXPECT_DINTERVAL_EQ(z1[0], (-5.0*y1[0]-z1[1]*x1[2])/x1[0]);
}


#endif



